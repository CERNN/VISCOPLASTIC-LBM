#include "hip/hip_runtime.h"
/*
*   @file parallelPlatesBounceBack.cu
*   @author Waine Jr. (waine@alunos.utfpr.edu.br)
*   @brief Parallel plates using bounce boundary conditions in walls,
*          periodic condition in flow direction and force in Z
*          N, S: Periodic; B, F: periodic; W, E: Wall
*   @version 0.3.0
*   @date 16/12/2019
*/

/*
*   LBM-CERNN
*   Copyright (C) 2018-2019 Waine Barbosa de Oliveira Junior
*
*   This program is free software; you can redistribute it and/or modify
*   it under the terms of the GNU General Public License as published by
*   the Free Software Foundation; either version 2 of the License, or
*   (at your option) any later version.
*
*   This program is distributed in the hope that it will be useful,
*   but WITHOUT ANY WARRANTY; without even the implied warranty of
*   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
*   GNU General Public License for more details.
*
*   You should have received a copy of the GNU General Public License along
*   with this program; if not, write to the Free Software Foundation, Inc.,
*   51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
*
*   Contact: cernn-ct@utfpr.edu.br and waine@alunos.utfpr.edu.br
*/

#include "boundaryConditionsBuilder.h"


__global__
void gpuBuildBoundaryConditions(NodeTypeMap* const gpuMapBC, int gpuNumber)
{
    const unsigned int x = threadIdx.x + blockDim.x * blockIdx.x;
    const unsigned int y = threadIdx.y + blockDim.y * blockIdx.y;
    const unsigned int z = threadIdx.z + blockDim.z * blockIdx.z;
    const unsigned int zDomain = z + NZ*gpuNumber;


    if(x >= NX || y >= NY || z >= NZ)
        return;

    gpuMapBC[idxScalar(x, y, z)].setIsUsed(true); //set all nodes fluid inicially and no bc
    gpuMapBC[idxScalar(x, y, z)].setSavePostCol(false); // set all nodes to not save post 
                                                    // collision population (just stream)
    gpuMapBC[idxScalar(x, y, z)].setSchemeBC(BC_NULL);
    gpuMapBC[idxScalar(x, y, z)].setGeometry(CONCAVE);
    gpuMapBC[idxScalar(x, y, z)].setUxIdx(0); // manually assigned (index of ux=0)
    gpuMapBC[idxScalar(x, y, z)].setUyIdx(0); // manually assigned (index of uy=0)
    gpuMapBC[idxScalar(x, y, z)].setUzIdx(0); // manually assigned (index of uz=0)
    gpuMapBC[idxScalar(x, y, z)].setRhoIdx(0); // manually assigned (index of rho=RHO_0)

    if (y == 0 && x == 0 && zDomain == 0) // SWB
    {
        gpuMapBC[idxScalar(x, y, z)].setSchemeBC(BC_SCHEME_BOUNCE_BACK);
        gpuMapBC[idxScalar(x, y, z)].setDirection(WEST);
    }
    else if (y == 0 && x == 0 && zDomain == (NZ_TOTAL-1)) // SWF
    {
        gpuMapBC[idxScalar(x, y, z)].setSchemeBC(BC_SCHEME_BOUNCE_BACK);
        gpuMapBC[idxScalar(x, y, z)].setDirection(WEST);
    }
    else if (y == 0 && x == (NX - 1) && zDomain == 0) // SEB
    {
        gpuMapBC[idxScalar(x, y, z)].setSchemeBC(BC_SCHEME_BOUNCE_BACK);
        gpuMapBC[idxScalar(x, y, z)].setDirection(EAST);
    }
    else if (y == 0 && x == (NX - 1) && zDomain == (NZ_TOTAL-1)) // SEF
    {
        gpuMapBC[idxScalar(x, y, z)].setSchemeBC(BC_SCHEME_BOUNCE_BACK);
        gpuMapBC[idxScalar(x, y, z)].setDirection(EAST);
    }
    else if (y == (NY - 1) && x == 0 && zDomain == 0) // NWB
    {
        gpuMapBC[idxScalar(x, y, z)].setSchemeBC(BC_SCHEME_BOUNCE_BACK);
        gpuMapBC[idxScalar(x, y, z)].setDirection(WEST);
    }
    else if (y == (NY - 1) && x == 0 && zDomain == (NZ_TOTAL-1)) // NWF
    {
        gpuMapBC[idxScalar(x, y, z)].setSchemeBC(BC_SCHEME_BOUNCE_BACK);
        gpuMapBC[idxScalar(x, y, z)].setDirection(WEST);
    }
    else if (y == (NY - 1) && x == (NX - 1) && zDomain == 0) // NEB
    {
        gpuMapBC[idxScalar(x, y, z)].setSchemeBC(BC_SCHEME_BOUNCE_BACK);
        gpuMapBC[idxScalar(x, y, z)].setDirection(EAST);

    }
    else if (y == (NY - 1) && x == (NX - 1) && zDomain == (NZ_TOTAL-1)) // NEF
    {
        gpuMapBC[idxScalar(x, y, z)].setSchemeBC(BC_SCHEME_BOUNCE_BACK);
        gpuMapBC[idxScalar(x, y, z)].setDirection(EAST);
    }
    else if (y == 0 && x == 0) // SW
    {
        gpuMapBC[idxScalar(x, y, z)].setSchemeBC(BC_SCHEME_BOUNCE_BACK);
        gpuMapBC[idxScalar(x, y, z)].setDirection(WEST);
    }
    else if (y == 0 && x == (NX - 1)) // SE
    {
        gpuMapBC[idxScalar(x, y, z)].setSchemeBC(BC_SCHEME_BOUNCE_BACK);
        gpuMapBC[idxScalar(x, y, z)].setDirection(EAST);
    }
    else if (y == (NY - 1) && x == 0) // NW
    {
        gpuMapBC[idxScalar(x, y, z)].setSchemeBC(BC_SCHEME_BOUNCE_BACK);
        gpuMapBC[idxScalar(x, y, z)].setDirection(WEST);
    }
    else if (y == (NY - 1) && x == (NX - 1)) // NE
    {
        gpuMapBC[idxScalar(x, y, z)].setSchemeBC(BC_SCHEME_BOUNCE_BACK);
        gpuMapBC[idxScalar(x, y, z)].setDirection(EAST);
    }
    else if (y == 0 && zDomain == 0) // SB
    {

    }
    else if (y == 0 && zDomain == (NZ_TOTAL-1)) // SF
    {

    }
    else if (y == (NY - 1) && zDomain == 0) // NB
    {

    }
    else if (y == (NY - 1) && zDomain == (NZ_TOTAL-1)) // NF
    {

    }
    else if (x == 0 && zDomain == 0) // WB
    {        
        gpuMapBC[idxScalar(x, y, z)].setSchemeBC(BC_SCHEME_BOUNCE_BACK);
        gpuMapBC[idxScalar(x, y, z)].setDirection(WEST);
    }
    else if (x == 0 && zDomain == (NZ_TOTAL-1)) // WF
    {
        gpuMapBC[idxScalar(x, y, z)].setSchemeBC(BC_SCHEME_BOUNCE_BACK);
        gpuMapBC[idxScalar(x, y, z)].setDirection(WEST);
    }
    else if (x == (NX - 1) && zDomain == 0) // EB
    {
        gpuMapBC[idxScalar(x, y, z)].setSchemeBC(BC_SCHEME_BOUNCE_BACK);
        gpuMapBC[idxScalar(x, y, z)].setDirection(EAST);
    }
    else if (x == (NX - 1) && zDomain == (NZ_TOTAL-1)) // EF
    {
        gpuMapBC[idxScalar(x, y, z)].setSchemeBC(BC_SCHEME_BOUNCE_BACK);
        gpuMapBC[idxScalar(x, y, z)].setDirection(EAST);
    }
    else if (y == 0) // S
    {

    }
    else if (y == (NY - 1)) // N
    {

    }
    else if (x == 0) // W
    {
        gpuMapBC[idxScalar(x, y, z)].setSchemeBC(BC_SCHEME_BOUNCE_BACK);
        gpuMapBC[idxScalar(x, y, z)].setDirection(WEST);
    }
    else if (x == (NX - 1)) // E
    {
        gpuMapBC[idxScalar(x, y, z)].setSchemeBC(BC_SCHEME_BOUNCE_BACK);
        gpuMapBC[idxScalar(x, y, z)].setDirection(EAST);
    }
    else if (zDomain == 0) // B
    {
    }
    else if (zDomain == (NZ_TOTAL-1)) // F
    {
    }
}


__device__
void gpuSchSpecial(NodeTypeMap* gpuNT, 
    dfloat* fPostStream,
    dfloat* fPostCol,
    const short unsigned int x, 
    const short unsigned int y, 
    const short unsigned int z)
{
    switch(gpuNT->getDirection())
    {
    case NORTH_WEST:
        // SPECIAL TREATMENT FOR NW
        break;

    case NORTH_EAST:
        // SPECIAL TREATMENT FOR NE
        break;

    case NORTH_FRONT:
        // SPECIAL TREATMENT FOR NF
        break;

    case NORTH_BACK:
        // SPECIAL TREATMENT FOR NB
        break;

    case SOUTH_WEST:
        // SPECIAL TREATMENT FOR SW
        break;

    case SOUTH_EAST:
        // SPECIAL TREATMENT FOR SE
        break;

    case SOUTH_FRONT:
        // SPECIAL TREATMENT FOR SF
        break;

    case SOUTH_BACK:
        // SPECIAL TREATMENT FOR SB
        break;

    case WEST_FRONT:
        // SPECIAL TREATMENT FOR WF
        break;

    case WEST_BACK:
        // SPECIAL TREATMENT FOR WB
        break;

    case EAST_FRONT:
        // SPECIAL TREATMENT FOR EF
        break;

    case EAST_BACK:
        // SPECIAL TREATMENT FOR EB
        break;

    case NORTH_WEST_FRONT:
        // SPECIAL TREATMENT FOR NWF
        break;

    case NORTH_WEST_BACK:
        // SPECIAL TREATMENT FOR NWB
        break;

    case NORTH_EAST_FRONT:
        // SPECIAL TREATMENT FOR NEF
        break;

    case NORTH_EAST_BACK:
        // SPECIAL TREATMENT FOR NEB
        break;

    case SOUTH_WEST_FRONT:
        // SPECIAL TREATMENT FOR SWF
        break;

    case SOUTH_WEST_BACK:
        // SPECIAL TREATMENT FOR SWB
        break;

    case SOUTH_EAST_FRONT:
        // SPECIAL TREATMENT FOR SEF
        break;

    case SOUTH_EAST_BACK:
        // SPECIAL TREATMENT FOR SEB
        break;
    
    default:
        break;
    }
}