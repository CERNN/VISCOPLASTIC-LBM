#include "hip/hip_runtime.h"
/*
*   @file lidDrivenBounceBack.cu
*   @author Waine Jr. (waine@alunos.utfpr.edu.br)
*   @brief Lid driven cavity with bounce back boundary conditions
*          N: ux=UMAX; W,E,S: wall; B,F: periodic
*   @version 0.2.0
*   @date 16/08/2019
*/

/*
*   LBM-CERNN
*   Copyright (C) 2018-2019 Waine Barbosa de Oliveira Junior
*
*   This program is free software; you can redistribute it and/or modify
*   it under the terms of the GNU General Public License as published by
*   the Free Software Foundation; either version 2 of the License, or
*   (at your option) any later version.
*
*   This program is distributed in the hope that it will be useful,
*   but WITHOUT ANY WARRANTY; without even the implied warranty of
*   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
*   GNU General Public License for more details.
*
*   You should have received a copy of the GNU General Public License along
*   with this program; if not, write to the Free Software Foundation, Inc.,
*   51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
*
*   Contact: cernn-ct@utfpr.edu.br and waine@alunos.utfpr.edu.br
*/

#include "boundaryConditionsBuilder.h"


__global__
void gpuBuildBoundaryConditions(NodeTypeMap* const gpuMapBC)
{
    const unsigned int x = threadIdx.x + blockDim.x * blockIdx.x;
    const unsigned int y = threadIdx.y + blockDim.y * blockIdx.y;
    const unsigned int z = threadIdx.z + blockDim.z * blockIdx.z;
    
    gpuMapBC[idxScalar(x, y, z)].setIsUsed(true); //set all nodes fluid inicially and no bc
    gpuMapBC[idxScalar(x, y, z)].setSchemeBC(BC_NULL);
    gpuMapBC[idxScalar(x, y, z)].setGeometry(CONCAVE);
    gpuMapBC[idxScalar(x, y, z)].setUxIdx(0); // manually assigned (index of ux=0)
    gpuMapBC[idxScalar(x, y, z)].setUyIdx(0); // manually assigned (index of uy=0)
    gpuMapBC[idxScalar(x, y, z)].setUzIdx(0); // manually assigned (index of uz=0)
    gpuMapBC[idxScalar(x, y, z)].setRhoIdx(0); // manually assigned (index of rho=RHO_0)

    if(y == 0 && x == 0 && z == 0) // SWB
    {
        gpuMapBC[idxScalar(x, y, z)].setSchemeBC(BC_SCHEME_BOUNCE_BACK);
        gpuMapBC[idxScalar(x, y, z)].setDirection(SOUTH_WEST);
    }
    else if(y == 0 && x == 0 && z == (NZ-1)) // SWF
    {
        gpuMapBC[idxScalar(x, y, z)].setSchemeBC(BC_SCHEME_BOUNCE_BACK);
        gpuMapBC[idxScalar(x, y, z)].setDirection(SOUTH_WEST);
    }
    else if(y == 0 && x == (NX-1) && z == 0) // SEB
    {
        gpuMapBC[idxScalar(x, y, z)].setSchemeBC(BC_SCHEME_BOUNCE_BACK);
        gpuMapBC[idxScalar(x, y, z)].setDirection(SOUTH_EAST);

    }
    else if(y == 0 && x == (NX-1) && z == (NZ-1)) // SEF
    {
        gpuMapBC[idxScalar(x, y, z)].setSchemeBC(BC_SCHEME_BOUNCE_BACK);
        gpuMapBC[idxScalar(x, y, z)].setDirection(SOUTH_EAST);
    }
    else if(y == (NY-1) && x == 0 && z == 0) // NWB
    {
        gpuMapBC[idxScalar(x, y, z)].setSchemeBC(BC_SCHEME_BOUNCE_BACK);
        gpuMapBC[idxScalar(x, y, z)].setDirection(NORTH_WEST);
    }
    else if(y == (NY-1) && x == 0 && z == (NZ-1)) // NWF
    {
        gpuMapBC[idxScalar(x, y, z)].setSchemeBC(BC_SCHEME_BOUNCE_BACK);
        gpuMapBC[idxScalar(x, y, z)].setDirection(NORTH_WEST);
    }
    else if(y == (NY-1) && x == (NX-1) && z == 0) // NEB
    {
        gpuMapBC[idxScalar(x, y, z)].setSchemeBC(BC_SCHEME_BOUNCE_BACK);
        gpuMapBC[idxScalar(x, y, z)].setDirection(NORTH_EAST);

    }
    else if(y == (NY-1) && x == (NX-1) && z == (NZ-1)) // NEF
    {
        gpuMapBC[idxScalar(x, y, z)].setSchemeBC(BC_SCHEME_BOUNCE_BACK);
        gpuMapBC[idxScalar(x, y, z)].setDirection(NORTH_EAST);
    }
    else if(y == 0 && x == 0) // SW
    {
        gpuMapBC[idxScalar(x, y, z)].setSchemeBC(BC_SCHEME_BOUNCE_BACK);
        gpuMapBC[idxScalar(x, y, z)].setDirection(SOUTH_WEST);
    }
    else if(y == 0 && x == (NX-1)) // SE
    {
        gpuMapBC[idxScalar(x, y, z)].setSchemeBC(BC_SCHEME_BOUNCE_BACK);
        gpuMapBC[idxScalar(x, y, z)].setDirection(SOUTH_EAST);
    }
    else if(y == (NY-1) && x == 0) // NW
    {
        gpuMapBC[idxScalar(x, y, z)].setSchemeBC(BC_SCHEME_BOUNCE_BACK);
        gpuMapBC[idxScalar(x, y, z)].setDirection(NORTH_WEST);
    }
    else if(y == (NY-1) && x == (NX-1)) // NE
    {
        gpuMapBC[idxScalar(x, y, z)].setSchemeBC(BC_SCHEME_BOUNCE_BACK);
        gpuMapBC[idxScalar(x, y, z)].setDirection(NORTH_EAST);
    }
    else if(y == 0 && z == 0) // SB
    {
        gpuMapBC[idxScalar(x, y, z)].setSchemeBC(BC_SCHEME_BOUNCE_BACK);
        gpuMapBC[idxScalar(x, y, z)].setDirection(SOUTH);
    }
    else if(y == 0 && z == (NZ-1)) // SF
    {
        gpuMapBC[idxScalar(x, y, z)].setSchemeBC(BC_SCHEME_BOUNCE_BACK);
        gpuMapBC[idxScalar(x, y, z)].setDirection(SOUTH);
    }
    else if(y == (NY-1) && z == 0) // NB
    {
        gpuMapBC[idxScalar(x, y, z)].setSchemeBC(BC_SCHEME_VEL_BOUNCE_BACK);
        gpuMapBC[idxScalar(x, y, z)].setDirection(NORTH);
        gpuMapBC[idxScalar(x, y, z)].setUxIdx(1); // manually assigned (index of ux=U_MAX)
    }
    else if(y == (NY-1) && z == (NZ-1)) // NF
    {
        gpuMapBC[idxScalar(x, y, z)].setSchemeBC(BC_SCHEME_VEL_BOUNCE_BACK);
        gpuMapBC[idxScalar(x, y, z)].setDirection(NORTH);
        gpuMapBC[idxScalar(x, y, z)].setUxIdx(1); // manually assigned (index of ux=U_MAX)
    }
    else if(x == 0 && z == 0) // WB
    {
        gpuMapBC[idxScalar(x, y, z)].setSchemeBC(BC_SCHEME_BOUNCE_BACK);
        gpuMapBC[idxScalar(x, y, z)].setDirection(WEST);
    }
    else if(x == 0 && z == (NZ-1)) // WF
    {
        gpuMapBC[idxScalar(x, y, z)].setSchemeBC(BC_SCHEME_BOUNCE_BACK);
        gpuMapBC[idxScalar(x, y, z)].setDirection(WEST);
    }
    else if(x == (NX-1) && z == 0) // EB
    {
        gpuMapBC[idxScalar(x, y, z)].setSchemeBC(BC_SCHEME_BOUNCE_BACK);
        gpuMapBC[idxScalar(x, y, z)].setDirection(EAST);
    }
    else if(x == (NX-1) && z == (NZ-1)) // EF
    {
        gpuMapBC[idxScalar(x, y, z)].setSchemeBC(BC_SCHEME_BOUNCE_BACK);
        gpuMapBC[idxScalar(x, y, z)].setDirection(EAST);
    }
    else if(y == 0) // S
    {
        gpuMapBC[idxScalar(x, y, z)].setSchemeBC(BC_SCHEME_BOUNCE_BACK);
        gpuMapBC[idxScalar(x, y, z)].setDirection(SOUTH);
    }
    else if(y == (NY-1)) // N
    {
        gpuMapBC[idxScalar(x, y, z)].setSchemeBC(BC_SCHEME_VEL_BOUNCE_BACK);
        gpuMapBC[idxScalar(x, y, z)].setDirection(NORTH);
        gpuMapBC[idxScalar(x, y, z)].setUxIdx(1); // manually assigned (index of ux=U_MAX)
    }
    else if(x == 0) // W
    {
        gpuMapBC[idxScalar(x, y, z)].setSchemeBC(BC_SCHEME_BOUNCE_BACK);
        gpuMapBC[idxScalar(x, y, z)].setDirection(WEST);
    }
    else if(x == (NX-1)) // E
    {
        gpuMapBC[idxScalar(x, y, z)].setSchemeBC(BC_SCHEME_BOUNCE_BACK);
        gpuMapBC[idxScalar(x, y, z)].setDirection(EAST);
    }
    else if(z == 0) // B
    {
    }
    else if(z == (NZ-1)) // F
    {
    }
}


__device__
void gpuSchSpecial(NodeTypeMap* gpuNT, 
    dfloat* f,
    dfloat* fNode, 
    const short unsigned int x, 
    const short unsigned int y, 
    const short unsigned int z)
{
    switch(gpuNT->getDirection())
    {
    case NORTH_WEST:
        // SPECIAL TREATMENT FOR NW
        break;

    case NORTH_EAST:
        // SPECIAL TREATMENT FOR NE
        break;

    case NORTH_FRONT:
        // SPECIAL TREATMENT FOR NF
        break;

    case NORTH_BACK:
        // SPECIAL TREATMENT FOR NB
        break;

    case SOUTH_WEST:
        // SPECIAL TREATMENT FOR SW
        break;

    case SOUTH_EAST:
        // SPECIAL TREATMENT FOR SE
        break;

    case SOUTH_FRONT:
        // SPECIAL TREATMENT FOR SF
        break;

    case SOUTH_BACK:
        // SPECIAL TREATMENT FOR SB
        break;

    case WEST_FRONT:
        // SPECIAL TREATMENT FOR WF
        break;

    case WEST_BACK:
        // SPECIAL TREATMENT FOR WB
        break;

    case EAST_FRONT:
        // SPECIAL TREATMENT FOR EF
        break;

    case EAST_BACK:
        // SPECIAL TREATMENT FOR EB
        break;

    case NORTH_WEST_FRONT:
        // SPECIAL TREATMENT FOR NWF
        break;

    case NORTH_WEST_BACK:
        // SPECIAL TREATMENT FOR NWB
        break;

    case NORTH_EAST_FRONT:
        // SPECIAL TREATMENT FOR NEF
        break;

    case NORTH_EAST_BACK:
        // SPECIAL TREATMENT FOR NEB
        break;

    case SOUTH_WEST_FRONT:
        // SPECIAL TREATMENT FOR SWF
        break;

    case SOUTH_WEST_BACK:
        // SPECIAL TREATMENT FOR SWB
        break;

    case SOUTH_EAST_FRONT:
        // SPECIAL TREATMENT FOR SEF
        break;

    case SOUTH_EAST_BACK:
        // SPECIAL TREATMENT FOR SEB
        break;
    
    default:
        break;
    }
}