#include "hip/hip_runtime.h"
#include "gInitialization.h"


__global__
void gpuInitializationG(
    gPopulations* gPop,
    Macroscopics* macr)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int z = threadIdx.z + blockDim.z * blockIdx.z;
    if (x >= NX || y >= NY || z >= NZ)
        return;

    size_t index = idxScalar(x, y, z);

    for (int i = 0; i < GQ; i++)
    {
        // calculate equilibrium population and initialize populations to equilibrium
        dfloat geq = gpu_g_eq(macr->G[index], macr->u.x[index],macr->u.y[index],macr->u.z[index],i);
        
        gPop->gPop[idxPop(x, y, z, i)] = geq;
        gPop->gPopAux[idxPop(x, y, z, i)] = geq;
    }
}