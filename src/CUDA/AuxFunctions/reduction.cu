#include "hip/hip_runtime.h"
/*
Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions
are met:
 * Redistributions of source code must retain the above copyright
   notice, this list of conditions and the following disclaimer.
 * Redistributions in binary form must reproduce the above copyright
   notice, this list of conditions and the following disclaimer in the
   documentation and/or other materials provided with the distribution.
 * Neither the name of NVIDIA CORPORATION nor the names of its
   contributors may be used to endorse or promote products derived
   from this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

For additional information on the license terms, see the CUDA EULA at
https://docs.nvidia.com/cuda/eula/index.html
*/


#include "reduction.cuh"


__global__ void reductionArray(float* g_idata, float* g_odata)
{
#if (BLOCK_LBM_SIZE == 512)
    __shared__ float sdata[BLOCK_LBM_SIZE];
#else
    extern __shared__ float sdata[];
#endif

    //global index in the array
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    //thread index in the array
    unsigned int tid = threadIdx.x;

    unsigned int bid = blockIdx.x + gridDim.x * (blockIdx.y + gridDim.y * (blockIdx.z));


    sdata[tid] = g_idata[i];
    //do reduction in shared mem
    for (unsigned int s = 1; s < blockDim.x; s *= 2) {
        if (tid % (2 * s) == 0) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0)
        g_odata[bid] = sdata[0];
}

__global__ void reductionMacro(float* g_idata, float* g_odata)
{
#if (BLOCK_LBM_SIZE == 512)
    __shared__ float sdata[BLOCK_LBM_SIZE];
#else
    extern __shared__ float sdata[];
#endif

    const short unsigned int x = threadIdx.x + blockDim.x * blockIdx.x;
    const short unsigned int y = threadIdx.y + blockDim.y * blockIdx.y;
    const short unsigned int z = threadIdx.z + blockDim.z * blockIdx.z;
    //global index in the array
    unsigned int i = idxScalar(x, y, z);

    //thread index in the array
    unsigned int tid = threadIdx.x;

    unsigned int bid = blockIdx.x + gridDim.x * (blockIdx.y + gridDim.y * (blockIdx.z));


    sdata[tid] = g_idata[i];
    //do reduction in shared mem
    for (unsigned int s = 1; s < blockDim.x; s *= 2) {
        if (tid % (2 * s) == 0) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0) 
        g_odata[bid] = sdata[0];
}