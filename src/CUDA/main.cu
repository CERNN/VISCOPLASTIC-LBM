/*
*   LBM-CERNN
*   Copyright (C) 2018-2019 Waine Barbosa de Oliveira Junior
*
*   This program is free software; you can redistribute it and/or modify
*   it under the terms of the GNU General Public License as published by
*   the Free Software Foundation; either version 2 of the License, or
*   (at your option) any later version.
*
*   This program is distributed in the hope that it will be useful,
*   but WITHOUT ANY WARRANTY; without even the implied warranty of
*   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
*   GNU General Public License for more details.
*
*   You should have received a copy of the GNU General Public License along
*   with this program; if not, write to the Free Software Foundation, Inc.,
*   51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
*
*   Contact: cernn-ct@utfpr.edu.br and waine@alunos.utfpr.edu.br
*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "treatData.h"
#include "lbmReport.h"
#include "lbm.h"
#include "lbmInitialization.h"
#include "boundaryConditionsBuilder.h"
#include "structs/boundaryConditionsInfo.h"


int main()
{
    // VARIABLE DECLARATIONS
    Populations* pop;
    Macroscopics* macr;
    Macroscopics macrCPUCurrent;
    Macroscopics macrCPUOld;
    MacrProc processData;
    BoundaryConditionsInfo* bcInfos;
    SimInfo info;
    float** randomNumbers = nullptr; // useful for turbulence
    int step = INI_STEP;
    dim3* gridsBC;

    // SETUP SAVING FOLDER
    folderSetup();

    // INITALIZE PROCESS DATA
    processData.step = &step;
    processData.macrCurr = &macrCPUCurrent;
    processData.macrOld = &macrCPUOld;
    
    // NUMBER OF DEVICES
    checkCudaErrors(hipGetDeviceCount(&info.numDevices));
    if(N_GPUS != info.numDevices){
            printf("N_GPUS is different than the number of detected GPUS\n");
            printf("N_GPUS: %d\n", N_GPUS);
            printf("Number of devices: %d\n", info.numDevices);
            return -1;
    }

    // ALLOCATION FOR CPU
    info.devices = (hipDeviceProp_t*) malloc(sizeof(hipDeviceProp_t)*N_GPUS);
    bcInfos = (BoundaryConditionsInfo*) malloc(sizeof(BoundaryConditionsInfo)*N_GPUS);
    gridsBC = (dim3*) malloc(sizeof(dim3)*N_GPUS);
    macrCPUCurrent.macrAllocation(IN_HOST);
    macrCPUOld.macrAllocation(IN_HOST);

    // STREAMS AND MEMORY ALLOCATION FOR GPU
    hipStream_t* streamsKernelLBM = (hipStream_t*) malloc(sizeof(hipStream_t)*N_GPUS); // stream kernel for each GPU
    checkCudaErrors(hipSetDevice(0));
    checkCudaErrors(hipMallocManaged((void**)&pop, 
        sizeof(Populations)*N_GPUS));
    checkCudaErrors(hipMallocManaged((void**)&macr, 
        sizeof(Macroscopics)*N_GPUS));
    checkCudaErrors(hipMallocManaged((void**)&randomNumbers, 
        sizeof(float*)*N_GPUS));

    // ALLOCATION AND CONFIGURATION FOR EACH GPU
    for(int i = 0; i < N_GPUS; i++)
    {
        checkCudaErrors(hipSetDevice(i));
        checkCudaErrors(hipGetDeviceProperties(&(info.devices[i]), i));
        checkCudaErrors(hipStreamCreate(&streamsKernelLBM[i]));
        pop[i].popAllocation();
        macr[i].macrAllocation(IN_VIRTUAL);
        if(RANDOM_NUMBERS)
        {
            checkCudaErrors(hipMallocManaged((void**)&randomNumbers[i], 
                sizeof(float)*numberNodes));
            initializationRandomNumbers(randomNumbers[i], CURAND_SEED);
            checkCudaErrors(hipDeviceSynchronize());
            getLastCudaError("random numbers transfer error");
        }
    }

/*
    ---------------------------------------------------------------------------
    ---------------------------------------------------------------------------
    ------------------ CODE BELOW DOES NOT SUPPORT MULTI GPU! -----------------
    ----------------------------- MUST BE UPDATED! ----------------------------
    ---------------------------------------------------------------------------
    ---------------------------------------------------------------------------
*/

    // GRID AND THREADS DEFINITION FOR LBM
    dim3 grid(((NX%nThreads)? (NX/nThreads+1) : (NX/nThreads)), NY, NZ);
    // threads in block
    dim3 threads(nThreads, 1, 1);

    // Grid and threads for memory transfers in multiGPUS
    dim3 gridTransfer(grid.x, grid.y, 1);
    dim3 threadsTransfer(nThreads, 1, 1);

    // REPORT
    printParamInfo(&info, true); fflush(stdout);
    printGPUInfo(&info); fflush(stdout);

    // BOUNDARY CONDITIONS INITIALIZATION
    for(int i = 0; i < N_GPUS; i++){
        checkCudaErrors(hipSetDevice(i));
        gpuBuildBoundaryConditions<<<grid, threads>>>(pop[i].mapBC, i);
    }

    checkCudaErrors(hipDeviceSynchronize());

    // Divide in two fors to allow kernels of "gpuBuilBoundaryConditions"
    // to run in parallel. Otherwise they would run sequentially
    for(int i = 0; i < N_GPUS; i++){
        bcInfos[i].setupBoundaryConditionsInfo(pop[i].mapBC);
    }

    // LBM INITIALIZATION
    // TODO: update initialization with files to multi GPU
    if(LOAD_POP)
    {
        FILE* filePop = fopen(STR_POP, "rb");
        if(filePop == nullptr)
        {
            printf("Error reading population file\n");
            return -1;
        }
        initializationPop(&pop[0], filePop);
        fclose (filePop);
        gpuUpdateMacr<<<grid, threads>>>(&pop[0], &macr[0]);
        checkCudaErrors(hipDeviceSynchronize());
        getLastCudaError("Update macroscopics error");
    }
    else 
    {
        if(LOAD_MACR)
        {   
            FILE* fileRho = fopen(STR_RHO, "rb");
            FILE* fileUx = fopen(STR_UX, "rb");
            FILE* fileUy = fopen(STR_UY, "rb");
            FILE* fileUz = fopen(STR_UZ, "rb");
            if(fileRho == nullptr || fileUz == nullptr 
                || fileUy == nullptr || fileUx == nullptr)
            {
                printf("Error reading macroscopics files\n");
                return -1;
            }
            initializationMacr(&macr[0], fileRho, fileUx, fileUy, fileUz);
            fclose (fileRho);
            fclose (fileUx);
            fclose (fileUy);
            fclose (fileUz);
        }
        
        for(int i = 0; i < N_GPUS; i++){
            checkCudaErrors(hipSetDevice(i));
            gpuInitialization<<<grid, threads>>>(&pop[i], &macr[i], LOAD_MACR, randomNumbers[i]);
        }
        checkCudaErrors(hipDeviceSynchronize());
        getLastCudaError("Initialization error");
    }

    // GRID AND THREAD DEFINITION FOR BOUNDARY CONDITIONS

    for(int i = 0; i < N_GPUS; i++)
        gridsBC[i] = dim3(((bcInfos[i].totalBCNodes%32)? (bcInfos[i].totalBCNodes/32+1) : 
                (bcInfos[i].totalBCNodes/32)), 1, 1); // TODO
    dim3 threadsBC(32, 1, 1);

    if(RANDOM_NUMBERS)
        checkCudaErrors(hipFree(randomNumbers));

    // TIMING
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    checkCudaErrors(hipEventRecord(start, 0));
    
    // LBM
    for(step = INI_STEP; step < N_STEPS; step++)
    {
        int aux = step-INI_STEP;
        // WHAT NEEDS TO BE DONE IN THIS TIME STEP
        bool save = false, rep = false;
        if(aux != 0)
        {
            if(MACR_SAVE != 0)
                save = !(aux % MACR_SAVE);
            if(DATA_REPORT != 0)
                rep = !(aux % DATA_REPORT);
        }

        // LBM SOLVER
        for(int i = 0; i < N_GPUS; i++){
            checkCudaErrors(hipSetDevice(i));
            gpuMacrCollisionStream<<<grid, threads, 0, streamsKernelLBM[i]>>>
                (pop[i].pop, pop[i].popAux, pop[i].mapBC, 
                &macr[i], rep || save || ((step+1)>=(int)N_STEPS), step);

            getLastCudaError("LBM kernel error\n");
        }

        if(N_GPUS > 1) {
            // Populations transfer
            for(int i = 0; i < N_GPUS; i++){
                checkCudaErrors(hipSetDevice(i));
                int nxt = (i+1)%NX;
                gpuPopulationsTransfer<<<gridTransfer, threadsTransfer, 0, streamsKernelLBM[i]>>>
                    (pop[i].pop, pop[i].popAux, pop[nxt].pop, pop[nxt].popAux);

                getLastCudaError("Mem transfer kernel error\n");
            }
        }

        // BOUNDARY CONDITIONS
        for(int i = 0; i < N_GPUS; i++){
            checkCudaErrors(hipSetDevice(i));
            checkCudaErrors(hipStreamSynchronize(streamsKernelLBM[i]));
            // BOUNDARY CONDITIONS
            if(bcInfos[i].totalBCNodes > 0){
                gpuApplyBC<<<gridsBC[i], threadsBC, 0, streamsKernelLBM[i]>>>
                    (pop[i].mapBC, pop[i].popAux, pop[i].pop, 
                    bcInfos[i].idxBCNodes, bcInfos[i].totalBCNodes);
                getLastCudaError("LBM kernel error\n");
            }
        }
        
        for(int i = 0; i < N_GPUS; i++){
            checkCudaErrors(hipStreamSynchronize(streamsKernelLBM[i]));
            pop[i].swapPop();
        }

        // SYNCHRONIZING
        if(save || rep)
        {
            printf("\n------------------------- Synchronizing in step %06d -------------------------\n", step); 
            fflush(stdout);
            checkCudaErrors(hipStreamSynchronize(streamsKernelLBM[0]));
            
            macrCPUOld.copyMacr(&macrCPUCurrent);        
            for(int i = 0; i < N_GPUS; i++){
                macrCPUCurrent.copyMacr(&macr[i], numberNodes*i);
            } 
        }

        // SAVE
        if(save)
        {
            printf("\n---------------------------- Saving in step %06d -----------------------------\n", step); 
            fflush(stdout);
            saveAllMacrBin(&macrCPUCurrent, step);
        }

        // REPORT
        if(rep)
        {
            treatData(&processData);
            printTreatData(&processData); 
            fflush(stdout);
            if(DATA_SAVE)
            {
                saveTreatData(&processData);
            }
            if(DATA_STOP)
            {
                if(stopSim(&processData))
                    break;
            }
        }
    }

    // TIMING
    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&(info.timeElapsed), start, stop));
    info.timeElapsed *= 0.001;

    // SAVE FINAL MACROSCOPICS
    
    for(int i = 0; i < N_GPUS; i++){
        macrCPUCurrent.copyMacr(&macr[i], numberNodes*i);
    } 
    saveAllMacrBin(&macrCPUCurrent, step);

    // SAVE FINAL POPULATIONS (IF REQUIRED)
    // TODO: update to multi GPU
    if(POP_SAVE)
        savePopBin(pop, step);

    // EVALUATE PERFORMANCE
    info.totalSteps = step - INI_STEP;
    size_t nodesUpdated = info.totalSteps * numberNodes * N_GPUS;
    info.MLUPS = (nodesUpdated / 1e6) / info.timeElapsed;
    // bandwidth for AB scheme and does not consider macroscopics transfers
    info.bandwidth = memSizePop*2.0*N_GPUS / (info.timeElapsed*BYTES_PER_GB) 
        * info.totalSteps;

    // SIMULATION INFO
    saveSimInfo(&info);

    // REPORT
    if(DATA_REPORT)
    {
        printTreatData(&processData);
        if(DATA_SAVE)
            saveTreatData(&processData);
    }
    printParamInfo(&info, true);
    printGPUInfo(&info);
    fflush(stdout);

    // FREE MEMORY FOR EACH GPU
    for(int i = 0; i < N_GPUS; i++)
    {
        checkCudaErrors(hipSetDevice(i));
        checkCudaErrors(hipStreamDestroy(streamsKernelLBM[i]));
        pop[i].popFree();
        macr[i].macrFree();
        bcInfos[i].freeIdxBC();
    }

    // FREE GPU VARIABLES
    checkCudaErrors(hipSetDevice(0));
    checkCudaErrors(hipFree(pop));
    checkCudaErrors(hipFree(macr));

    // FREE CPU VARIABLES
    macrCPUCurrent.macrFree();
    macrCPUOld.macrFree();
    free(info.devices);
    free(bcInfos);
    free(gridsBC);

    return 0;
}