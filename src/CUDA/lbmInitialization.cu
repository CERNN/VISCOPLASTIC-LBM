#include "hip/hip_runtime.h"
/*
*   LBM-CERNN
*   Copyright (C) 2018-2019 Waine Barbosa de Oliveira Junior
*
*   This program is free software; you can redistribute it and/or modify
*   it under the terms of the GNU General Public License as published by
*   the Free Software Foundation; either version 2 of the License, or
*   (at your option) any later version.
*
*   This program is distributed in the hope that it will be useful,
*   but WITHOUT ANY WARRANTY; without even the implied warranty of
*   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
*   GNU General Public License for more details.
*
*   You should have received a copy of the GNU General Public License along
*   with this program; if not, write to the Free Software Foundation, Inc.,
*   51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
*
*   Contact: cernn-ct@utfpr.edu.br and waine@alunos.utfpr.edu.br
*/

#include "lbmInitialization.h"


__host__
void initializationPop( 
    Populations* pop,
    FILE* filePop,
    FILE* filePopAux)
{
    dfloat* tmp = (dfloat*)malloc(totalMemSizePop);
    fread(tmp, totalMemSizePop, 1, filePop);

    for(int i = 0; i < N_GPUS; i++){
        size_t base_idx = numberNodes*Q*i;
        checkCudaErrors(hipMemcpy(pop[i].pop, tmp+base_idx, memSizePop, hipMemcpyDefault));
    }

    fread(tmp, totalMemSizePop, 1, filePopAux);

    for(int i = 0; i < N_GPUS; i++){
        size_t base_idx = numberNodes*Q*i;
        checkCudaErrors(hipMemcpy(pop[i].popAux, tmp+base_idx, memSizePop, hipMemcpyDefault));
    }

    free(tmp);
}


__host__
void initializationMacr(
    Macroscopics* macr,
    FILE* fileRho,
    FILE* fileUx,
    FILE* fileUy,
    FILE* fileUz)
{
    dfloat* tmp = (dfloat*)malloc(totalMemSizeScalar);

    fread(tmp, totalMemSizeScalar, 1, fileRho);
    checkCudaErrors(hipMemcpy(macr->rho, tmp, totalMemSizeScalar, hipMemcpyDefault));

    fread(tmp, totalMemSizeScalar, 1, fileUx);
    checkCudaErrors(hipMemcpy(macr->ux, tmp, totalMemSizeScalar, hipMemcpyDefault));

    fread(tmp, totalMemSizeScalar, 1, fileUy);
    checkCudaErrors(hipMemcpy(macr->uy, tmp, totalMemSizeScalar, hipMemcpyDefault));

    fread(tmp, totalMemSizeScalar, 1, fileUz);
    checkCudaErrors(hipMemcpy(macr->uz, tmp, totalMemSizeScalar, hipMemcpyDefault));

    free(tmp);
}


__host__
void initializationRandomNumbers(
    float* randomNumbers, int seed)
{
    hiprandGenerator_t gen;

    // Create pseudo-random number generator
    checkCurandStatus(hiprandCreateGenerator(&gen,
        HIPRAND_RNG_PSEUDO_DEFAULT));
    
    // Set generator seed
    checkCurandStatus(hiprandSetPseudoRandomGeneratorSeed(gen,
        CURAND_SEED));
    
    // Generate NX*NY*NZ floats on device, using normal distribution
    // with mean=0 and std_dev=NORMAL_STD_DEV
    checkCurandStatus(hiprandGenerateNormal(gen, randomNumbers, numberNodes,
        0, CURAND_STD_DEV));

    checkCurandStatus(hiprandDestroyGenerator(gen));
}


__global__
void gpuInitialization(
    Populations pop,
    Macroscopics macr,
    bool isMacrInit,
    float* randomNumbers)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int z = threadIdx.z + blockDim.z * blockIdx.z;
    if (x >= NX || y >= NY || z >= NZ)
        return;

    size_t index = idxScalar(x, y, z);

    if (!isMacrInit)
    {
        gpuMacrInitValue(&macr, randomNumbers, x, y, z);
    }

    for (int i = 0; i < Q; i++)
    {
        // calculate equilibrium population and initialize populations to equilibrium
        dfloat feq = gpu_f_eq(w[i] * macr.rho[index],
            3 * (macr.ux[index] * cx[i] + macr.uy[index] * cy[i] + macr.uz[index] * cz[i]),
            1 - 1.5*(  macr.ux[index] * macr.ux[index] 
                 + macr.uy[index] * macr.uy[index] 
                 + macr.uz[index] * macr.uz[index]));
        
        pop.pop[idxPop(x, y, z, i)] = feq;
        pop.popAux[idxPop(x, y, z, i)] = feq;
    }
}


__device__
void gpuMacrInitValue(
    Macroscopics* macr,
    float* randomNumbers,
    int x, int y, int z)
{
    macr->rho[idxScalar(x, y, z)] = RHO_0;
    macr->ux[idxScalar(x, y, z)] = 0;
    macr->uy[idxScalar(x, y, z)] = 0;
    macr->uz[idxScalar(x, y, z)] = 0;

    // Example of usage of random numbers for turbulence in parallel plates flow in z

    /*
    dfloat y_visc = 6.59, ub_f = 15.6, uc_f = 18.2;
​
    // logaritimic velocity profile
    dfloat uz_log, pos = (y < NY/2 ? y + 0.5 : NY - (y + 0.5));
    uz_log = (uc_f*U_TAU)*(pos/del)*(pos/del);
​
    macr->uz[idxScalar(x, y, z)] = uz_log;
    macr->ux[idxScalar(x, y, z)] = 0.0;
    macr->uy[idxScalar(x, y, z)] = 0.0;
    macr->rho[idxScalar(x, y, z)] = RHO_0;
​
    // perturbation
    dfloat pert = 0.1;
    int l = idxScalar(x, y, z), Nt = numberNodes;
    macr->uz[idxScalar(x, y, z)] += (ub_f*U_TAU)*pert*randomNumbers[l + NZ - Nt*((l + NZ) / Nt)];
    macr->ux[idxScalar(x, y, z)] += (ub_f*U_TAU)*pert*randomNumbers[l + NX - Nt*((l + NX) / Nt)];
    macr->uy[idxScalar(x, y, z)] += (ub_f*U_TAU)*pert*randomNumbers[l + NY - Nt*((l + NY) / Nt)];
    */
}
