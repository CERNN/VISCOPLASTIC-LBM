#include "hip/hip_runtime.h"
/*
*   LBM-CERNN
*   Copyright (C) 2018-2019 Waine Barbosa de Oliveira Junior
*
*   This program is free software; you can redistribute it and/or modify
*   it under the terms of the GNU General Public License as published by
*   the Free Software Foundation; either version 2 of the License, or
*   (at your option) any later version.
*
*   This program is distributed in the hope that it will be useful,
*   but WITHOUT ANY WARRANTY; without even the implied warranty of
*   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
*   GNU General Public License for more details.
*
*   You should have received a copy of the GNU General Public License along
*   with this program; if not, write to the Free Software Foundation, Inc.,
*   51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
*
*   Contact: cernn-ct@utfpr.edu.br and waine@alunos.utfpr.edu.br
*/

#include "lbmInitialization.h"


__host__
void initializationRandomNumbers(
    float* randomNumbers, int seed)
{
    hiprandGenerator_t gen;

    // Create pseudo-random number generator
    checkCurandStatus(hiprandCreateGenerator(&gen,
        HIPRAND_RNG_PSEUDO_DEFAULT));
    
    // Set generator seed
    checkCurandStatus(hiprandSetPseudoRandomGeneratorSeed(gen,
        CURAND_SEED));
    
    // Generate NX*NY*NZ floats on device, using normal distribution
    // with mean=0 and std_dev=NORMAL_STD_DEV
    checkCurandStatus(hiprandGenerateNormal(gen, randomNumbers, NUMBER_LBM_NODES,
        0, CURAND_STD_DEV));

    checkCurandStatus(hiprandDestroyGenerator(gen));
}


__global__
void gpuInitialization(
    Populations pop,
    Macroscopics macr,
    float* randomNumbers)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int z = threadIdx.z + blockDim.z * blockIdx.z;
    if (x >= NX || y >= NY || z >= NZ)
        return;

    size_t index = idxScalar(x, y, z+MACR_BORDER_NODES);

    gpuMacrInitValue(&macr, randomNumbers, x, y, z);

    for (int i = 0; i < Q; i++)
    {
        // calculate equilibrium population and initialize populations to equilibrium
        dfloat feq = gpu_f_eq(w[i] * macr.rho[index],
            3 * (macr.u.x[index] * cx[i] + macr.u.y[index] * cy[i] + macr.u.z[index] * cz[i]),
            1 - 1.5*(  macr.u.x[index] * macr.u.x[index] 
                 + macr.u.y[index] * macr.u.y[index] 
                 + macr.u.z[index] * macr.u.z[index]));
        
        pop.pop[idxPop(x, y, z, i)] = feq;
        pop.popAux[idxPop(x, y, z, i)] = feq;
    }
}


__device__
void gpuMacrInitValue(
    Macroscopics* macr,
    float* randomNumbers,
    int x, int y, int z)
{
    // +MACR_BORDER_NODES because of the ghost nodes
    macr->rho[idxScalar(x, y, z+MACR_BORDER_NODES)] = RHO_0;
    macr->u.x[idxScalar(x, y, z+MACR_BORDER_NODES)] = 0;
    macr->u.y[idxScalar(x, y, z+MACR_BORDER_NODES)] = 0;
    macr->u.z[idxScalar(x, y, z+MACR_BORDER_NODES)] = 0;

    #ifdef IBM
    macr->f.x[idxScalar(x, y, z+MACR_BORDER_NODES)] = FX;
    macr->f.y[idxScalar(x, y, z+MACR_BORDER_NODES)] = FY;
    macr->f.z[idxScalar(x, y, z+MACR_BORDER_NODES)] = FZ;
    #endif
    #ifdef NON_NEWTONIAN_FLUID
    macr->omega[idxScalar(x, y, z)] = 0;
    #endif

    // Example of usage of random numbers for turbulence in parallel plates flow in z

    /*
    dfloat y_visc = 6.59, ub_f = 15.6, uc_f = 18.2;
​
    // logaritimic velocity profile
    dfloat uz_log, pos = (y < NY/2 ? y + 0.5 : NY - (y + 0.5));
    uz_log = (uc_f*U_TAU)*(pos/del)*(pos/del);
​
    macr->u.z[idxScalar(x, y, z)] = uz_log;
    macr->u.x[idxScalar(x, y, z)] = 0.0;
    macr->u.y[idxScalar(x, y, z)] = 0.0;
    macr->rho[idxScalar(x, y, z)] = RHO_0;
​
    // perturbation
    dfloat pert = 0.1;
    int l = idxScalar(x, y, z), Nt = NUMBER_LBM_NODES;
    macr->u.z[idxScalar(x, y, z)] += (ub_f*U_TAU)*pert*randomNumbers[l + NZ - Nt*((l + NZ) / Nt)];
    macr->u.x[idxScalar(x, y, z)] += (ub_f*U_TAU)*pert*randomNumbers[l + NX - Nt*((l + NX) / Nt)];
    macr->u.y[idxScalar(x, y, z)] += (ub_f*U_TAU)*pert*randomNumbers[l + NY - Nt*((l + NY) / Nt)];
    */
}
