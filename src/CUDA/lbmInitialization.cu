#include "hip/hip_runtime.h"
/*
*   LBM-CERNN
*   Copyright (C) 2018-2019 Waine Barbosa de Oliveira Junior
*
*   This program is free software; you can redistribute it and/or modify
*   it under the terms of the GNU General Public License as published by
*   the Free Software Foundation; either version 2 of the License, or
*   (at your option) any later version.
*
*   This program is distributed in the hope that it will be useful,
*   but WITHOUT ANY WARRANTY; without even the implied warranty of
*   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
*   GNU General Public License for more details.
*
*   You should have received a copy of the GNU General Public License along
*   with this program; if not, write to the Free Software Foundation, Inc.,
*   51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
*
*   Contact: cernn-ct@utfpr.edu.br and waine@alunos.utfpr.edu.br
*/

#include "lbmInitialization.h"


__host__
void initializationPop( 
    Populations* pop,
    FILE* filePop)
{
    dfloat* tmp = (dfloat*)malloc(memSizePop);
    if (filePop != NULL)
    {
        fread(tmp, memSizePop, 1, filePop);
        checkCudaErrors(hipMemcpy(pop->pop, tmp, memSizePop, hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(pop->popAux, tmp, memSizePop, hipMemcpyHostToDevice));
    }
    free(tmp);
}


__host__
void initializationMacr(
    Macroscopics* macr,
    FILE* fileRho,
    FILE* fileUx,
    FILE* fileUy,
    FILE* fileUz)
{
    dfloat* tmp = (dfloat*)malloc(memSizeScalar);
    if (fileRho != NULL)
    {
        fread(tmp, memSizeScalar, 1, fileRho);
        checkCudaErrors(hipMemcpy(macr->rho, tmp, memSizeScalar, hipMemcpyHostToDevice));
    }
    if (fileUx != NULL)
    {
        fread(tmp, memSizeScalar, 1, fileUx);
        checkCudaErrors(hipMemcpy(macr->ux, tmp, memSizeScalar, hipMemcpyHostToDevice));
    }
    if (fileUy != NULL)
    {
        fread(tmp, memSizeScalar, 1, fileUy);
        checkCudaErrors(hipMemcpy(macr->uy, tmp, memSizeScalar, hipMemcpyHostToDevice));
    }
    if (fileUz != NULL)
    {
        fread(tmp, memSizeScalar, 1, fileUz);
        checkCudaErrors(hipMemcpy(macr->uz, tmp, memSizeScalar, hipMemcpyHostToDevice));
    }
    free(tmp);
}


__global__
void gpuInitialization(
    Populations* pop,
    Macroscopics* macr,
    bool isMacrInit)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int z = threadIdx.z + blockDim.z * blockIdx.z;
    if (x >= NX || y >= NY || z >= NZ)
        return;

    size_t index = idxScalar(x, y, z);

    if (!isMacrInit)
    {
        gpuMacrInitValue(macr, x, y, z);
    }

    for (int i = 0; i < Q; i++)
    {
        // calculate equilibrium population and initialize populations to equilibrium
        dfloat feq = gpu_f_eq(w[i] * macr->rho[index],
            3 * (macr->ux[index] * cx[i] + macr->uy[index] * cy[i] + macr->uz[index] * cz[i]),
            1 - (  macr->ux[index] * macr->ux[index] 
                 + macr->uy[index] * macr->uy[index] 
                 + macr->uz[index] * macr->uz[index]));
        
        pop->pop[idxPop(x, y, z, i)] = feq;
        pop->popAux[idxPop(x, y, z, i)] = feq;
    }
}


__device__
void gpuMacrInitValue(
    Macroscopics* macr,
    int x, int y, int z)
{
    macr->rho[idxScalar(x, y, z)] = RHO_0;
    macr->ux[idxScalar(x, y, z)] = 0;
    macr->uy[idxScalar(x, y, z)] = 0;
    macr->uz[idxScalar(x, y, z)] = 0;
}
