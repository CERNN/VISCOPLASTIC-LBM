#include "hip/hip_runtime.h"
/*
*   LBM-CERNN
*   Copyright (C) 2018-2019 Waine Barbosa de Oliveira Junior
*
*   This program is free software; you can redistribute it and/or modify
*   it under the terms of the GNU General Public License as published by
*   the Free Software Foundation; either version 2 of the License, or
*   (at your option) any later version.
*
*   This program is distributed in the hope that it will be useful,
*   but WITHOUT ANY WARRANTY; without even the implied warranty of
*   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
*   GNU General Public License for more details.
*
*   You should have received a copy of the GNU General Public License along
*   with this program; if not, write to the Free Software Foundation, Inc.,
*   51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
*
*   Contact: cernn-ct@utfpr.edu.br and waine@alunos.utfpr.edu.br
*/

#include "lbmInitialization.h"


__host__
void initializationPop( 
    Populations* pop,
    FILE* filePop,
    FILE* filePopAux)
{
    dfloat* tmp = (dfloat*)malloc(TOTAL_MEM_SIZE_POP);
    fread(tmp, TOTAL_MEM_SIZE_POP, 1, filePop);

    for(int i = 0; i < N_GPUS; i++){
        size_t base_idx = NUMBER_LBM_NODES*Q*i;
        checkCudaErrors(hipMemcpy(pop[i].pop, tmp+base_idx, MEM_SIZE_POP, hipMemcpyDefault));
    }

    fread(tmp, TOTAL_MEM_SIZE_POP, 1, filePopAux);

    for(int i = 0; i < N_GPUS; i++){
        size_t base_idx = NUMBER_LBM_NODES*Q*i;
        checkCudaErrors(hipMemcpy(pop[i].popAux, tmp+base_idx, MEM_SIZE_POP, hipMemcpyDefault));
    }

    free(tmp);
}


__host__
void initializationMacr(
    Macroscopics* macr,
    FILE* fileRho,
    FILE* fileUx,
    FILE* fileUy,
    FILE* fileUz,
    FILE* fileFx,
    FILE* fileFy,
    FILE* fileFz,
    FILE* fileOmega)
{
    dfloat* tmp = (dfloat*)malloc(TOTAL_MEM_SIZE_SCALAR);

    fread(tmp, TOTAL_MEM_SIZE_SCALAR, 1, fileRho);
    checkCudaErrors(hipMemcpy(macr->rho, tmp, TOTAL_MEM_SIZE_SCALAR, hipMemcpyDefault));

    fread(tmp, TOTAL_MEM_SIZE_SCALAR, 1, fileUx);
    checkCudaErrors(hipMemcpy(macr->u.x, tmp, TOTAL_MEM_SIZE_SCALAR, hipMemcpyDefault));

    fread(tmp, TOTAL_MEM_SIZE_SCALAR, 1, fileUy);
    checkCudaErrors(hipMemcpy(macr->u.y, tmp, TOTAL_MEM_SIZE_SCALAR, hipMemcpyDefault));

    fread(tmp, TOTAL_MEM_SIZE_SCALAR, 1, fileUz);
    checkCudaErrors(hipMemcpy(macr->u.z, tmp, TOTAL_MEM_SIZE_SCALAR, hipMemcpyDefault));

    #ifdef IBM
    fread(tmp, TOTAL_MEM_SIZE_SCALAR, 1, fileFx);
    checkCudaErrors(hipMemcpy(macr->f.x, tmp, TOTAL_MEM_SIZE_SCALAR, hipMemcpyDefault));

    fread(tmp, TOTAL_MEM_SIZE_SCALAR, 1, fileFy);
    checkCudaErrors(hipMemcpy(macr->f.y, tmp, TOTAL_MEM_SIZE_SCALAR, hipMemcpyDefault));

    fread(tmp, TOTAL_MEM_SIZE_SCALAR, 1, fileFz);
    checkCudaErrors(hipMemcpy(macr->f.z, tmp, TOTAL_MEM_SIZE_SCALAR, hipMemcpyDefault));
    #endif

    #ifdef NON_NEWTONIAN_FLUID
    fread(tmp, TOTAL_MEM_SIZE_SCALAR, 1, fileOmega);
    checkCudaErrors(hipMemcpy(macr->omega, tmp, TOTAL_MEM_SIZE_SCALAR, hipMemcpyDefault));
    #endif

    free(tmp);
}


__host__
void initializationRandomNumbers(
    float* randomNumbers, int seed)
{
    hiprandGenerator_t gen;

    // Create pseudo-random number generator
    checkCurandStatus(hiprandCreateGenerator(&gen,
        HIPRAND_RNG_PSEUDO_DEFAULT));
    
    // Set generator seed
    checkCurandStatus(hiprandSetPseudoRandomGeneratorSeed(gen,
        CURAND_SEED));
    
    // Generate NX*NY*NZ floats on device, using normal distribution
    // with mean=0 and std_dev=NORMAL_STD_DEV
    checkCurandStatus(hiprandGenerateNormal(gen, randomNumbers, NUMBER_LBM_NODES,
        0, CURAND_STD_DEV));

    checkCurandStatus(hiprandDestroyGenerator(gen));
}


__global__
void gpuInitialization(
    Populations pop,
    Macroscopics macr,
    bool isMacrInit,
    float* randomNumbers)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int z = threadIdx.z + blockDim.z * blockIdx.z;
    if (x >= NX || y >= NY || z >= NZ)
        return;

    size_t index = idxScalar(x, y, z);

    if (!isMacrInit)
    {
        gpuMacrInitValue(&macr, randomNumbers, x, y, z);
    }

    for (int i = 0; i < Q; i++)
    {
        // calculate equilibrium population and initialize populations to equilibrium
        dfloat feq = gpu_f_eq(w[i] * macr.rho[index],
            3 * (macr.u.x[index] * cx[i] + macr.u.y[index] * cy[i] + macr.u.z[index] * cz[i]),
            1 - 1.5*(  macr.u.x[index] * macr.u.x[index] 
                 + macr.u.y[index] * macr.u.y[index] 
                 + macr.u.z[index] * macr.u.z[index]));
        
        pop.pop[idxPop(x, y, z, i)] = feq;
        pop.popAux[idxPop(x, y, z, i)] = feq;
    }
}


__device__
void gpuMacrInitValue(
    Macroscopics* macr,
    float* randomNumbers,
    int x, int y, int z)
{
    macr->rho[idxScalar(x, y, z)] = RHO_0;
    macr->u.x[idxScalar(x, y, z)] = 0;
    macr->u.y[idxScalar(x, y, z)] = 0;
    macr->u.z[idxScalar(x, y, z)] = 0;

    #ifdef IBM
    macr->f.x[idxScalar(x, y, z)] = FX;
    macr->f.y[idxScalar(x, y, z)] = FY;
    macr->f.z[idxScalar(x, y, z)] = FZ;
    #endif
    #ifdef NON_NEWTONIAN_FLUID
    macr->omega[idxScalar(x, y, z)] = 0;
    #endif

    // Example of usage of random numbers for turbulence in parallel plates flow in z

    /*
    dfloat y_visc = 6.59, ub_f = 15.6, uc_f = 18.2;
​
    // logaritimic velocity profile
    dfloat uz_log, pos = (y < NY/2 ? y + 0.5 : NY - (y + 0.5));
    uz_log = (uc_f*U_TAU)*(pos/del)*(pos/del);
​
    macr->u.z[idxScalar(x, y, z)] = uz_log;
    macr->u.x[idxScalar(x, y, z)] = 0.0;
    macr->u.y[idxScalar(x, y, z)] = 0.0;
    macr->rho[idxScalar(x, y, z)] = RHO_0;
​
    // perturbation
    dfloat pert = 0.1;
    int l = idxScalar(x, y, z), Nt = NUMBER_LBM_NODES;
    macr->u.z[idxScalar(x, y, z)] += (ub_f*U_TAU)*pert*randomNumbers[l + NZ - Nt*((l + NZ) / Nt)];
    macr->u.x[idxScalar(x, y, z)] += (ub_f*U_TAU)*pert*randomNumbers[l + NX - Nt*((l + NX) / Nt)];
    macr->u.y[idxScalar(x, y, z)] += (ub_f*U_TAU)*pert*randomNumbers[l + NY - Nt*((l + NY) / Nt)];
    */
}
