#include "hip/hip_runtime.h"
#include "lbm.h"

__global__ 
void gpuMacrCollisionStream(
    dfloat* const pop,
    dfloat* const popAux,
    NodeTypeMap* const mapBC,
    Macroscopics const macr,
    bool const save,
    int const step)
{
    const short unsigned int x = threadIdx.x + blockDim.x * blockIdx.x;
    const short unsigned int y = threadIdx.y + blockDim.y * blockIdx.y;
    const short unsigned int z = threadIdx.z + blockDim.z * blockIdx.z;
    if (x >= NX || y >= NY || z >= NZ)
        return;
    if(!mapBC[idxScalar(x, y, z)].getIsUsed())
        return;

    // Adjacent coordinates
    const unsigned short int xp1 = (x + 1) % NX;
    const unsigned short int yp1 = (y + 1) % NY;
    const unsigned short int zp1 = (z + 1) % NZ;
    const unsigned short int xm1 = (NX + x - 1) % NX;
    const unsigned short int ym1 = (NY + y - 1) % NY;
    const unsigned short int zm1 = (NZ + z - 1) % NZ;

    // Node populations
    dfloat fNode[Q];

    // Load populations
    #pragma unroll
    for (char i = 0; i < Q; i++)
        fNode[i] = pop[idxPop(x, y, z, i)];

    // Calculate macroscopics
    // rho = sum(f[i])
    // ux = (sum(f[i]*cx[i])+0.5*FX) / rho
    // uy = (sum(f[i]*cy[i])+0.5*FY) / rho
    // uz = (sum(f[i]*cz[i])+0.5*FZ) / rho
    #ifdef D3Q19
    const dfloat rhoVar = fNode[0] + fNode[1] + fNode[2] + fNode[3] + fNode[4] 
        + fNode[5] + fNode[6] + fNode[7] + fNode[8] + fNode[9] + fNode[10] 
        + fNode[11] + fNode[12] + fNode[13] + fNode[14] + fNode[15] + fNode[16] 
        + fNode[17] + fNode[18];
    const dfloat invRho = 1/rhoVar;
    const dfloat uxVar = ((fNode[1] + fNode[7] + fNode[9] + fNode[13] + fNode[15])
        - (fNode[2] + fNode[8] + fNode[10] + fNode[14] + fNode[16]) + 0.5*FX) * invRho;
    const dfloat uyVar = ((fNode[3] + fNode[7] + fNode[11] + fNode[14] + fNode[17])
        - (fNode[4] + fNode[8] + fNode[12] + fNode[13] + fNode[18]) + 0.5*FY) * invRho;
    const dfloat uzVar = ((fNode[5] + fNode[9] + fNode[11] + fNode[16] + fNode[18])
        - (fNode[6] + fNode[10] + fNode[12] + fNode[15] + fNode[17]) + 0.5*FZ) * invRho;
    #endif // !D3Q19
    #ifdef D3Q27
    const dfloat rhoVar = fNode[0] + fNode[1] + fNode[2] + fNode[3] + fNode[4] 
        + fNode[5] + fNode[6] + fNode[7] + fNode[8] + fNode[9] + fNode[10] 
        + fNode[11] + fNode[12] + fNode[13] + fNode[14] + fNode[15] + fNode[16] 
        + fNode[17] + fNode[18] + fNode[19] + fNode[20] + fNode[21] + fNode[22]
        + fNode[23] + fNode[24] + fNode[25] + fNode[26];
        const dfloat invRho = 1/rhoVar;
    const dfloat uxVar = ((fNode[1] + fNode[7] + fNode[9] + fNode[13] + fNode[15]
        + fNode[19] + fNode[21] + fNode[23] + fNode[26]) 
        - (fNode[2] + fNode[8] + fNode[10] + fNode[14] + fNode[16] + fNode[20]
        + fNode[22] + fNode[24] + fNode[25]) + 0.5*FX) * invRho;
    const dfloat uyVar = ((fNode[3] + fNode[7] + fNode[11] + fNode[14] + fNode[17]
        + fNode[19] + fNode[21] + fNode[24] + fNode[25])
        - (fNode[4] + fNode[8] + fNode[12] + fNode[13] + fNode[18] + fNode[20]
        + fNode[22] + fNode[23] + fNode[26]) + 0.5*FY) * invRho;
    const dfloat uzVar = ((fNode[5] + fNode[9] + fNode[11] + fNode[16] + fNode[18]
        + fNode[19] + fNode[22] + fNode[23] + fNode[25])
        - (fNode[6] + fNode[10] + fNode[12] + fNode[15] + fNode[17] + fNode[20]
        + fNode[21] + fNode[24] + fNode[26]) + 0.5*FZ) * invRho;
    #endif // !D3Q27

    if (save)
    {
        macr.rho[idxScalar(x, y, z)] = rhoVar;
        macr.ux[idxScalar(x, y, z)] = uxVar;
        macr.uy[idxScalar(x, y, z)] = uyVar;
        macr.uz[idxScalar(x, y, z)] = uzVar;
    }

    // Calculate temporary variables
    const dfloat p1_muu15 = 1 - 1.5 * (uxVar * uxVar + 
        uyVar * uyVar + uzVar * uzVar);
    const dfloat rhoW0 = rhoVar * W0;
    const dfloat rhoW1 = rhoVar * W1;
    const dfloat rhoW2 = rhoVar * W2;
    const dfloat W1t9d2 = W1 * 4.5;
    const dfloat W2t9d2 = W2 * 4.5;
    #ifdef D3Q27
    const dfloat rhoW3 = rhoVar * W3;
    const dfloat W3t9d2 = W3 * 4.5;
    #endif
    const dfloat ux3 = 3 * uxVar;
    const dfloat uy3 = 3 * uyVar;
    const dfloat uz3 = 3 * uzVar;
    const dfloat ux3ux3d2 = ux3*ux3*0.5;
    const dfloat ux3uy3 = ux3*uy3;
    const dfloat ux3uz3 = ux3*uz3;
    const dfloat uy3uy3d2 = uy3*uy3*0.5;
    const dfloat uy3uz3 = uy3*uz3;
    const dfloat uz3uz3d2 = uz3*uz3*0.5;

    // Terms to use to recursive calculations
    #ifdef D3Q19
    dfloat terms[6];
    #endif 
    #ifdef D3Q27
    dfloat terms[11];
    #endif
    dfloat multiplyTerm = 1;
    dfloat auxTerm;


    // Calculate pineq(alfa, beta)
    // pineqAB = pops - popsEquilibrium
    #ifdef D3Q19
    const dfloat pineqXX = (fNode[1] + fNode[2] + fNode[7] + fNode[8] + fNode[9] 
            + fNode[10] + fNode[13] + fNode[14] + fNode[15] + fNode[16]) -
            (2*rhoW1*(p1_muu15 + ux3ux3d2) + 
             4*rhoW2*(2*p1_muu15 + 2*ux3ux3d2 + uy3uy3d2 + uz3uz3d2));
    const dfloat pineqYY = (fNode[3] + fNode[4] + fNode[7] + fNode[8] + fNode[11]
            + fNode[12] + fNode[13] + fNode[14] + fNode[17] + fNode[18]) -
            (2*rhoW1*(p1_muu15 + uy3uy3d2) + 
             4*rhoW2*(2*p1_muu15 + ux3ux3d2 + 2*uy3uy3d2 + uz3uz3d2)); 
    const dfloat pineqZZ = (fNode[5] + fNode[6] + fNode[9] + fNode[10] + fNode[11]
            + fNode[12] + fNode[15] + fNode[16] + fNode[17] + fNode[18]) -
            (2*rhoW1*(p1_muu15 + uz3uz3d2) + 
             4*rhoW2*(2*p1_muu15 + ux3ux3d2 + uy3uy3d2 + 2*uz3uz3d2));
    const dfloat pineqXYt2 = ((fNode[7] + fNode[8] - fNode[13] - fNode[14]) - 
            (4*rhoW2*(ux3uy3))) * 2;
    const dfloat pineqXZt2 = ((fNode[9] + fNode[10] - fNode[15] - fNode[16]) -
            (4*rhoW2*(ux3uz3))) * 2;
    const dfloat pineqYZt2 = ((fNode[11] + fNode[12] - fNode[17] - fNode[18]) -
            (4*rhoW2*(uy3uz3))) * 2;
    #endif // !D3Q19 
    #ifdef D3Q27
    const dfloat aux = (fNode[19] + fNode[20] + fNode[21] + fNode[22] + fNode[23]
            + fNode[24] + fNode[25] + fNode[26]) - 
            (8*rhoW3*(ux3ux3d2 + uy3uy3d2 + uz3uz3d2));
    const dfloat pineqXX = (fNode[1] + fNode[2] + fNode[7] + fNode[8] + fNode[9] 
            + fNode[10] + fNode[13] + fNode[14] + fNode[15] + fNode[16] + aux) -
            (2*rhoW1*(p1_muu15 + ux3ux3d2) + 
             4*rhoW2*(2*p1_muu15 + 2*ux3ux3d2 + uy3uy3d2 + uz3uz3d2));
    const dfloat pineqYY = (fNode[3] + fNode[4] + fNode[7] + fNode[8] + fNode[11]
            + fNode[12] + fNode[13] + fNode[14] + fNode[17] + fNode[18] + aux) -
            (2*rhoW1*(p1_muu15 + uy3uy3d2) + 
             4*rhoW2*(ux3ux3d2 + 2*uy3uy3d2 + uz3uz3d2)); 
    const dfloat pineqZZ = (fNode[5] + fNode[6] + fNode[9] + fNode[10] + fNode[11]
            + fNode[12] + fNode[15] + fNode[16] + fNode[17] + fNode[18] + aux) -
            (2*rhoW1*(p1_muu15 + uz3uz3d2) + 
             4*rhoW2*(ux3ux3d2 + uy3uy3d2 + 2*uz3uz3d2));
    const dfloat pineqXYt2 = ((fNode[7] + fNode[8] - fNode[13] - fNode[14] + fNode[19]
            + fNode[20] + fNode[21] + fNode[22] - fNode[23] - fNode[24] - fNode[25]
            - fNode[26]) - 
            (4*rhoW2*(ux3uy3) + 8*rhoW3*(ux3uy3))) * 2;
    const dfloat pineqXZt2 = ((fNode[9] + fNode[10] - fNode[15] - fNode[16] + fNode[19]
            + fNode[20] - fNode[21] - fNode[22] + fNode[23] + fNode[24] - fNode[25]
            - fNode[26]) - 
            (4*rhoW2*(ux3uz3) + 8*rhoW3*(ux3uz3))) * 2;
    const dfloat pineqYZt2 = ((fNode[11] + fNode[12] - fNode[17] - fNode[18] + fNode[19]
            + fNode[20] - fNode[21] - fNode[22] - fNode[23] - fNode[24] + fNode[25]
            + fNode[26]) - 
            (4*rhoW2*(uy3uz3) + 8*rhoW3*(uy3uz3))) * 2;
    #endif // !D3Q27

    // Calculate regularization terms 
    // terms[i] = Q[i, alfa, beta]*pi[i, alfa, beta] - c[i, alfa]*F[alfa]/3
    // terms[0] -> population 0
    // terms[1] -> population 1
    // terms[2] -> population 2
    // terms[3] -> population 3
    // terms[4] -> population 4
    terms[0] = -pineqXX/3 - pineqYY/3 - pineqZZ/3;
    terms[1] = terms[0] + (-FX_D3 + pineqXX);
    terms[2] = terms[0] + ( FX_D3 + pineqXX);
    terms[3] = terms[0] + (-FY_D3 + pineqYY);
    terms[4] = terms[0] + ( FY_D3 + pineqYY);
    #ifdef D3Q27
    // terms[5] -> population 7
    // terms[6] -> population 8
    // terms[7] -> population 9
    // terms[8] -> population 10
    // terms[9] -> population 11
    // terms[10] -> population 12
    terms[5] = terms[1] + (-FY_D3 + pineqXYt2 + pineqYY);
    terms[6] = terms[2] + ( FY_D3 + pineqXYt2 + pineqYY);
    terms[7] = terms[1] + (-FZ_D3 + pineqXZt2 + pineqZZ);
    terms[8] = terms[2] + ( FZ_D3 + pineqXZt2 + pineqZZ);
    terms[9] = terms[3] + (-FZ_D3 + pineqYZt2 + pineqZZ);
    terms[10] = terms[4] + ( FZ_D3 + pineqYZt2 + pineqZZ);
    #endif
    
    // Calculate regularized population to fNode
    // fNode[i] = 4.5*w[i](Q[i, alfa, beta]*pi[i, alfa, beta] 
    //          - c[i, alfa]*F[alfa]/3)
    multiplyTerm = W0*4.5;
    fNode[0] = multiplyTerm*terms[0];
    multiplyTerm = W1t9d2;
    fNode[1] = multiplyTerm*terms[1];
    fNode[2] = multiplyTerm*terms[2];
    fNode[3] = multiplyTerm*terms[3];
    fNode[4] = multiplyTerm*terms[4];
    fNode[5] = multiplyTerm*(terms[0] + (-FZ_D3 + pineqZZ));
    fNode[6] = multiplyTerm*(terms[0] + ( FZ_D3 + pineqZZ));
    multiplyTerm = W2t9d2;
    fNode[7] = multiplyTerm*(terms[1] + (-FY_D3 + pineqXYt2 + pineqYY));
    fNode[8] = multiplyTerm*(terms[2] + ( FY_D3 + pineqXYt2 + pineqYY));
    fNode[9] = multiplyTerm*(terms[1] + (-FZ_D3 + pineqXZt2 + pineqZZ));
    fNode[10] = multiplyTerm*(terms[2] + ( FZ_D3 + pineqXZt2 + pineqZZ));
    fNode[11] = multiplyTerm*(terms[3] + (-FZ_D3 + pineqYZt2 + pineqZZ));
    fNode[12] = multiplyTerm*(terms[4] + ( FZ_D3 + pineqYZt2 + pineqZZ));
    fNode[13] = multiplyTerm*(terms[1] + ( FY_D3 - pineqXYt2 + pineqYY));
    fNode[14] = multiplyTerm*(terms[2] + (-FY_D3 - pineqXYt2 + pineqYY));
    fNode[15] = multiplyTerm*(terms[1] + ( FZ_D3 - pineqXZt2 + pineqZZ));
    fNode[16] = multiplyTerm*(terms[2] + (-FZ_D3 - pineqXZt2 + pineqZZ));
    fNode[17] = multiplyTerm*(terms[3] + ( FZ_D3 - pineqYZt2 + pineqZZ));
    fNode[18] = multiplyTerm*(terms[4] + (-FZ_D3 - pineqYZt2 + pineqZZ));
    #ifdef D3Q27
    multiplyTerm = W3t9d2;
    fNode[19] = multiplyTerm*(terms[5] + (-FZ_D3 + pineqXZt2 + pineqYZt2 + pineqZZ));
    fNode[20] = multiplyTerm*(terms[6] + ( FZ_D3 + pineqXZt2 + pineqYZt2 + pineqZZ));
    fNode[21] = multiplyTerm*(terms[5] + ( FZ_D3 - pineqXZt2 - pineqYZt2 + pineqZZ));
    fNode[22] = multiplyTerm*(terms[6] + (-FZ_D3 - pineqXZt2 - pineqYZt2 + pineqZZ));
    fNode[23] = multiplyTerm*(terms[7] + ( FY_D3 - pineqXYt2 + pineqYY - pineqYZt2));
    fNode[24] = multiplyTerm*(terms[8] + (-FY_D3 - pineqXYt2 + pineqYY - pineqYZt2));
    fNode[25] = multiplyTerm*(terms[9] + ( FX_D3 + pineqXX - pineqXYt2 - pineqXZt2));
    fNode[26] = multiplyTerm*(terms[10] + (-FX_D3 + pineqXX - pineqXYt2 - pineqXZt2));
    #endif

    // Collision to fNode:
    // fNode = (1 - 1/TAU)*f1 + fEq + (1 - 0.5/TAU)*force ->
    // fNode = (1 - OMEGA)*f1 + fEq + (1 - 0.5*0MEGA)*force->
    // fNode = T_OMEGA * f1 + fEq + TT_OMEGA*force

    // Sequence is:
    // fNode *= T_OMEGA
    // fNode += fEq
    // fNode += TT_OMEGA*force

    #pragma unroll
    for(char i = 0; i < Q; i++)
        fNode[i] *= T_OMEGA;

    // Calculate equilibrium terms 
    // terms = 0.5*uc3^2 + uc3
    // terms[0] -> population 0
    // terms[1] -> population 1
    // terms[2] -> population 2
    // terms[3] -> population 3
    // terms[4] -> population 4
    terms[0] = p1_muu15;
    terms[1] = terms[0] + ( ux3 + ux3ux3d2);
    terms[2] = terms[0] + (-ux3 + ux3ux3d2);
    terms[3] = terms[0] + ( uy3 + uy3uy3d2);
    terms[4] = terms[0] + (-uy3 + uy3uy3d2);
    #ifdef D3Q27
    // terms[5] -> population 7
    // terms[6] -> population 8
    // terms[7] -> population 9
    // terms[8] -> population 10
    // terms[9] -> population 11
    // terms[10] -> population 12
    terms[5] = terms[1] + ( uy3 + ux3uy3 + uy3uy3d2);
    terms[6] = terms[2] + (-uy3 + ux3uy3 + uy3uy3d2);
    terms[7] = terms[1] + ( uz3 + ux3uz3 + uz3uz3d2);
    terms[8] = terms[2] + (-uz3 + ux3uz3 + uz3uz3d2);
    terms[9] = terms[3] + ( uz3 + uy3uz3 + uz3uz3d2);
    terms[10] = terms[4] + (-uz3 + uy3uz3 + uz3uz3d2);
    #endif

    // fNode += fEq
    multiplyTerm = rhoW0;
    fNode[0] += multiplyTerm*terms[0];
    multiplyTerm = rhoW1;
    fNode[1] += multiplyTerm*terms[1];
    fNode[2] += multiplyTerm*terms[2];
    fNode[3] += multiplyTerm*terms[3];
    fNode[4] += multiplyTerm*terms[4];
    fNode[5] += multiplyTerm*(terms[0] + ( uz3 + uz3uz3d2));
    fNode[6] += multiplyTerm*(terms[0] + (-uz3 + uz3uz3d2));
    multiplyTerm = rhoW2;
    fNode[7]  += multiplyTerm*(terms[1] + ( uy3 + ux3uy3 + uy3uy3d2));
    fNode[8]  += multiplyTerm*(terms[2] + (-uy3 + ux3uy3 + uy3uy3d2));
    fNode[9]  += multiplyTerm*(terms[1] + ( uz3 + ux3uz3 + uz3uz3d2));
    fNode[10] += multiplyTerm*(terms[2] + (-uz3 + ux3uz3 + uz3uz3d2));
    fNode[11] += multiplyTerm*(terms[3] + ( uz3 + uy3uz3 + uz3uz3d2));
    fNode[12] += multiplyTerm*(terms[4] + (-uz3 + uy3uz3 + uz3uz3d2));
    fNode[13] += multiplyTerm*(terms[1] + (-uy3 - ux3uy3 + uy3uy3d2));
    fNode[14] += multiplyTerm*(terms[2] + ( uy3 - ux3uy3 + uy3uy3d2));
    fNode[15] += multiplyTerm*(terms[1] + (-uz3 - ux3uz3 + uz3uz3d2));
    fNode[16] += multiplyTerm*(terms[2] + ( uz3 - ux3uz3 + uz3uz3d2));
    fNode[17] += multiplyTerm*(terms[3] + (-uz3 - uy3uz3 + uz3uz3d2));
    fNode[18] += multiplyTerm*(terms[4] + ( uz3 - uy3uz3 + uz3uz3d2));
    #ifdef D3Q27
    multiplyTerm = rhoW3;
    fNode[19] += multiplyTerm*(terms[5] + ( uz3 + ux3uz3 + uy3uz3 + uz3uz3d2));
    fNode[20] += multiplyTerm*(terms[6] + (-uz3 + ux3uz3 + uy3uz3 + uz3uz3d2));
    fNode[21] += multiplyTerm*(terms[5] + (-uz3 - ux3uz3 - uy3uz3 + uz3uz3d2));
    fNode[22] += multiplyTerm*(terms[6] + ( uz3 - ux3uz3 - uy3uz3 + uz3uz3d2));
    fNode[23] += multiplyTerm*(terms[7] + (-uy3 - ux3uy3 + uy3uy3d2 - uy3uz3));
    fNode[24] += multiplyTerm*(terms[8] + ( uy3 - ux3uy3 + uy3uy3d2 - uy3uz3));
    fNode[25] += multiplyTerm*(terms[9] + (-ux3 + ux3ux3d2 - ux3uy3 - ux3uz3));
    fNode[26] += multiplyTerm*(terms[10] + ( ux3 + ux3ux3d2 - ux3uy3 - ux3uz3));
    #endif

    // calculate force term
    // term[0] -> population 0
    // term[1] -> population 1
    // term[2] -> population 3
    // term[3] -> population 7
    // term[4] -> population 9
    // term[5] -> population 11
    terms[0] = - FX*ux3 - FY*uy3 - FZ*uz3;
    terms[1] = terms[0] + (FX*( 3*ux3 + 3));
    terms[2] = terms[0] + (FY*( 3*uy3 + 3));
    terms[3] = terms[1] + (FX*( 3*uy3) + FY*( 3*ux3 + 3*uy3 + 3));
    terms[4] = terms[1] + (FX*( 3*uz3) + FZ*( 3*ux3 + 3*uz3 + 3));
    terms[5] = terms[2] + (FY*( 3*uz3) + FZ*( 3*uy3 + 3*uz3 + 3));
    #ifdef D3Q27
    // term[6] -> population 19
    terms[6] = terms[3] + (FX*( 3*uz3) + FY*( 3*uz3) + FZ*( 3*ux3 + 3*uy3 + 3*uz3 + 3));
    #endif

    // fNode += TT_OMEGA * force
    multiplyTerm = W0*TT_OMEGA;
    fNode[0] += multiplyTerm*terms[0];
    multiplyTerm = W1*TT_OMEGA;
    fNode[1] += multiplyTerm*terms[1];
    fNode[2] += multiplyTerm*(terms[1] + (FX*(-6)));
    fNode[3] += multiplyTerm*terms[2];
    fNode[4] += multiplyTerm*(terms[2] + (FY*(-6)));
    auxTerm = terms[0] + (FZ*( 3*uz3 + 3));
    fNode[5] += multiplyTerm*auxTerm;
    fNode[6] += multiplyTerm*(auxTerm + (FZ*(-6)));
    multiplyTerm = W2*TT_OMEGA;
    fNode[7] += multiplyTerm*terms[3];
    fNode[8] += multiplyTerm*(terms[3] + (FX*(-6) + FY*(-6)));
    fNode[9] += multiplyTerm*terms[4];
    fNode[10] += multiplyTerm*(terms[4] + (FX*(-6) + FZ*(-6)));
    fNode[11] += multiplyTerm*(terms[5]);
    fNode[12] += multiplyTerm*(terms[5] + (FY*(-6) + FZ*(-6)));
    auxTerm = terms[3] + (FX*(-6*uy3) + FY*(-6*ux3 - 6));
    fNode[13] += multiplyTerm*(auxTerm);
    fNode[14] += multiplyTerm*(auxTerm + (FX*(-6) + FY*( 6)));
    auxTerm = terms[4] + (FX*(-6*uz3) + FZ*(-6*ux3 - 6));
    fNode[15] += multiplyTerm*auxTerm;
    fNode[16] += multiplyTerm*(auxTerm + (FX*(-6) + FZ*( 6)));
    auxTerm = terms[5] + (FY*(-6*uz3) + FZ*(-6*uy3 - 6));
    fNode[17] += multiplyTerm*auxTerm;
    fNode[18] += multiplyTerm*(auxTerm + (FY*(-6) + FZ*( 6)));
    #ifdef D3Q27
    multiplyTerm = W3*TT_OMEGA;
    fNode[19] += multiplyTerm*terms[6];
    fNode[20] += multiplyTerm*(terms[6] + (FX*(-6) + FY*(-6) + FZ*(-6)));
    auxTerm = terms[6] + (FX*(-6*uz3) + FY*(-6*uz3) + FZ*(-6*ux3 - 6*uy3 - 6));
    fNode[21] += multiplyTerm*auxTerm;
    fNode[22] += multiplyTerm*(auxTerm + (FX*(-6) + FY*(-6) + FZ*( 6)));
    auxTerm = terms[6] + (FX*(-6*uy3) + FY*(-6*ux3 - 6*uz3 - 6) + FZ*(-6*uy3));
    fNode[23] += multiplyTerm*auxTerm;
    fNode[24] += multiplyTerm*(auxTerm + (FX*(-6) + FY*( 6) + FZ*(-6)));
    auxTerm = terms[6] + (FX*(-6*uy3 - 6*uz3 - 6) + FY*(-6*ux3) + FZ*(-6*ux3));
    fNode[25] += multiplyTerm*auxTerm;
    fNode[26] += multiplyTerm*(auxTerm + (FX*( 6) + FY*(-6) + FZ*(-6)));
    #endif

    // Save post collision populations of boundary conditions nodes
    if(mapBC[idxScalar(x, y, z)].getSavePostCol())  
    {
        #pragma unroll
        for (char i = 0; i < Q; i++)
            pop[idxPop(x, y, z, i)] = fNode[i];
    }

    // Streaming to popAux
    // popAux(x+cx, y+cy, z+cz, i) = pop(x, y, z, i) 
    // The populations that shoudn't be streamed will be changed by the boundary conditions
    popAux[idxPop(x, y, z, 0)] = fNode[0];
    popAux[idxPop(xp1, y, z, 1)] = fNode[1];
    popAux[idxPop(xm1, y, z, 2)] = fNode[2];
    popAux[idxPop(x, yp1, z, 3)] = fNode[3];
    popAux[idxPop(x, ym1, z, 4)] = fNode[4];
    popAux[idxPop(x, y, zp1, 5)] = fNode[5];
    popAux[idxPop(x, y, zm1, 6)] = fNode[6];
    popAux[idxPop(xp1, yp1, z, 7)] = fNode[7];
    popAux[idxPop(xm1, ym1, z, 8)] = fNode[8];
    popAux[idxPop(xp1, y, zp1, 9)] = fNode[9];
    popAux[idxPop(xm1, y, zm1, 10)] = fNode[10];
    popAux[idxPop(x, yp1, zp1, 11)] = fNode[11];
    popAux[idxPop(x, ym1, zm1, 12)] = fNode[12];
    popAux[idxPop(xp1, ym1, z, 13)] = fNode[13];
    popAux[idxPop(xm1, yp1, z, 14)] = fNode[14];
    popAux[idxPop(xp1, y, zm1, 15)] = fNode[15];
    popAux[idxPop(xm1, y, zp1, 16)] = fNode[16];
    popAux[idxPop(x, yp1, zm1, 17)] = fNode[17];
    popAux[idxPop(x, ym1, zp1, 18)] = fNode[18];
    #ifdef D3Q27
    popAux[idxPop(xp1, yp1, zp1, 19)] = fNode[19];
    popAux[idxPop(xm1, ym1, zm1, 20)] = fNode[20];
    popAux[idxPop(xp1, yp1, zm1, 21)] = fNode[21];
    popAux[idxPop(xm1, ym1, zp1, 22)] = fNode[22];
    popAux[idxPop(xp1, ym1, zp1, 23)] = fNode[23];
    popAux[idxPop(xm1, yp1, zm1, 24)] = fNode[24];
    popAux[idxPop(xm1, yp1, zp1, 25)] = fNode[25];
    popAux[idxPop(xp1, ym1, zm1, 26)] = fNode[26];
    #endif
}


__global__
void gpuUpdateMacr(
    Populations* pop,
    Macroscopics* macr)
{
    const unsigned int z = threadIdx.z + blockDim.z * blockIdx.z;
    const unsigned int y = threadIdx.y + blockDim.y * blockIdx.y;
    const unsigned int x = threadIdx.x + blockDim.x * blockIdx.x;
    if (x >= NX || y >= NY || z >= NZ)
        return;

    // load populations
    dfloat fNode[Q];
    for (unsigned char i = 0; i < Q; i++)
        fNode[i] = pop->pop[idxPop(x, y, z, i)];

    // calc for macroscopics
    // rho = sum(f[i])
    // ux = sum(f[i]*cx[i]) / rho
    // uy = sum(f[i]*cy[i]) / rho
    // uz = sum(f[i]*cz[i]) / rho
    #ifdef D3Q19
    const dfloat rhoVar = fNode[0] + fNode[1] + fNode[2] + fNode[3] + fNode[4] 
        + fNode[5] + fNode[6] + fNode[7] + fNode[8] + fNode[9] + fNode[10] 
        + fNode[11] + fNode[12] + fNode[13] + fNode[14] + fNode[15] + fNode[16] 
        + fNode[17] + fNode[18];
    const dfloat invRho = 1/rhoVar;
    const dfloat uxVar = ((fNode[1] + fNode[7] + fNode[9] + fNode[13] + fNode[15])
        - (fNode[2] + fNode[8] + fNode[10] + fNode[14] + fNode[16]) + 0.5*FX) * invRho;
    const dfloat uyVar = ((fNode[3] + fNode[7] + fNode[11] + fNode[14] + fNode[17])
        - (fNode[4] + fNode[8] + fNode[12] + fNode[13] + fNode[18]) + 0.5*FY) * invRho;
    const dfloat uzVar = ((fNode[5] + fNode[9] + fNode[11] + fNode[16] + fNode[18])
        - (fNode[6] + fNode[10] + fNode[12] + fNode[15] + fNode[17]) + 0.5*FZ) * invRho;
    #endif // !D3Q19
    #ifdef D3Q27
    const dfloat rhoVar = fNode[0] + fNode[1] + fNode[2] + fNode[3] + fNode[4] 
        + fNode[5] + fNode[6] + fNode[7] + fNode[8] + fNode[9] + fNode[10] 
        + fNode[11] + fNode[12] + fNode[13] + fNode[14] + fNode[15] + fNode[16] 
        + fNode[17] + fNode[18] + fNode[19] + fNode[20] + fNode[21] + fNode[22]
        + fNode[23] + fNode[24] + fNode[25] + fNode[26];
        const dfloat invRho = 1/rhoVar;
    const dfloat uxVar = ((fNode[1] + fNode[7] + fNode[9] + fNode[13] + fNode[15]
        + fNode[19] + fNode[21] + fNode[23] + fNode[26]) 
        - (fNode[2] + fNode[8] + fNode[10] + fNode[14] + fNode[16] + fNode[20]
        + fNode[22] + fNode[24] + fNode[25]) + 0.5*FX) * invRho;
    const dfloat uyVar = ((fNode[3] + fNode[7] + fNode[11] + fNode[14] + fNode[17]
        + fNode[19] + fNode[21] + fNode[24] + fNode[25])
        - (fNode[4] + fNode[8] + fNode[12] + fNode[13] + fNode[18] + fNode[20]
        + fNode[22] + fNode[23] + fNode[26]) + 0.5*FY) * invRho;
    const dfloat uzVar = ((fNode[5] + fNode[9] + fNode[11] + fNode[16] + fNode[18]
        + fNode[19] + fNode[22] + fNode[23] + fNode[25])
        - (fNode[6] + fNode[10] + fNode[12] + fNode[15] + fNode[17] + fNode[20]
        + fNode[21] + fNode[24] + fNode[26]) + 0.5*FZ) * invRho;
    #endif // !D3Q27
    macr->rho[idxScalar(x, y, z)] = rhoVar;
    macr->ux[idxScalar(x, y, z)] = uxVar;
    macr->uy[idxScalar(x, y, z)] = uyVar;
    macr->uz[idxScalar(x, y, z)] = uzVar;
}


__global__
void gpuApplyBC(NodeTypeMap* mapBC,  
    dfloat* popPostStream,
    dfloat* popPostCol,
    size_t* idxsBCNodes,
    size_t totalBCNodes)
{
    const unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;

    if(i >= totalBCNodes)
        return;
    // converts 1D index to 3D location
    const size_t idx = idxsBCNodes[i];
    const unsigned int x = idx % NX;
    const unsigned int y = (idx/NX) % NY;
    const unsigned int z = idx/(NX*NY);

    gpuBoundaryConditions(&(mapBC[idx]), popPostStream, popPostCol, x, y, z);
}

__global__
void gpuPopulationsTransfer(
    dfloat* popPostStreamBase,
    dfloat* popPostCollBase,
    dfloat* popPostStreamNxt,
    dfloat* popPostCollNxt)
{
    const unsigned short int x = threadIdx.x + blockDim.x * blockIdx.x;
    const unsigned short int y = threadIdx.y + blockDim.y * blockIdx.y;
    const unsigned short int zMax = NZ-1;

    if (x >= NX || y >= NY)
        return;

    // This takes into account that the populations are "teleported"
    // from one side of domain to another. So the population with cz=-1
    // in z = 0 is streamed to z = NZ-1.
    // In this way, to retrieve a population that should have been sent 
    // to the adjacent node, but was "teleported", the part of the domain 
    // to which it was streamed must be read.
    // Also important to notice is that the popBase is above the popNext,
    // so the lower level of popBase must be streamed to the higher level of
    // popNext and vice versa

    // pop[6] -> cz = 1; pop[5] -> cz = -1;
    popPostStreamBase[idxPop(x, y, 0, 5)] = popPostStreamNxt[idxPop(x, y, 0, 5)];
    popPostStreamNxt[idxPop(x, y, zMax, 6)] = popPostStreamBase[idxPop(x, y, zMax, 6)];

    // pop[9] -> cz = 1; pop[10] -> cz = -1;
    popPostStreamBase[idxPop(x, y, 0, 9)] = popPostStreamNxt[idxPop(x, y, 0, 9)];
    popPostStreamNxt[idxPop(x, y, zMax, 10)] = popPostStreamBase[idxPop(x, y, zMax, 10)];
    
    // pop[11] -> cz = 1; pop[12] -> cz = -1;
    popPostStreamBase[idxPop(x, y, 0, 11)] = popPostStreamNxt[idxPop(x, y, 0, 11)];
    popPostStreamNxt[idxPop(x, y, zMax, 12)] = popPostStreamBase[idxPop(x, y, zMax, 12)];
    
    // pop[15] -> cz = 1; pop[16] -> cz = -1;
    popPostStreamBase[idxPop(x, y, 0, 16)] = popPostStreamNxt[idxPop(x, y, 0, 16)];
    popPostStreamNxt[idxPop(x, y, zMax, 15)] = popPostStreamBase[idxPop(x, y, zMax, 15)];

    // pop[18] -> cz = 1; pop[17] -> cz = -1;
    popPostStreamBase[idxPop(x, y, 0, 18)] =   popPostStreamNxt[idxPop(x, y, 0, 18)];
    popPostStreamNxt[idxPop(x, y, zMax, 17)] = popPostStreamBase[idxPop(x, y, zMax, 17)];

    #ifdef D3Q27
    // pop[19] -> cz = 1; pop[20] -> cz = -1;
    popPostStreamBase[idxPop(x, y, 0, 19)] = popPostStreamNxt[idxPop(x, y, 0, 19)];
    popPostStreamNxt[idxPop(x, y, zMax, 20)] = popPostStreamBase[idxPop(x, y, zMax, 20)];

    // pop[22] -> cz = 1; pop[21] -> cz = -1;
    popPostStreamBase[idxPop(x, y, 0, 22)] = popPostStreamNxt[idxPop(x, y, 0, 22)];
    popPostStreamNxt[idxPop(x, y, zMax, 21)] = popPostStreamBase[idxPop(x, y, zMax, 21)];

    // pop[23] -> cz = 1; pop[24] -> cz = -1;
    popPostStreamBase[idxPop(x, y, 0, 23)] = popPostStreamNxt[idxPop(x, y, 0, 23)];
    popPostStreamNxt[idxPop(x, y, zMax, 24)] = popPostStreamBase[idxPop(x, y, zMax, 24)];

    // pop[25] -> cz = 1; pop[26] -> cz = -1;
    popPostStreamBase[idxPop(x, y, 0, 25)] = popPostStreamNxt[idxPop(x, y, 0, 25)];
    popPostStreamNxt[idxPop(x, y, zMax, 26)] = popPostStreamBase[idxPop(x, y, zMax, 26)];
    #endif
}