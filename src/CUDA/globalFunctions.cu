#include "hip/hip_runtime.h"
#include "globalFunctions.h"

__global__
void copyFromArray(dfloat3SoA dst, dfloat3SoA src){
    const unsigned int x = threadIdx.x + blockDim.x * blockIdx.x;
    const unsigned int y = threadIdx.y + blockDim.y * blockIdx.y;
    const unsigned int z = threadIdx.z + blockDim.z * blockIdx.z;
    if (x >= NX || y >= NY || z >= (NZ+4))
        return;

    size_t i = idxScalar(x, y, z);

    dst.x[i] = src.x[i];
    dst.y[i] = src.y[i];
    dst.z[i] = src.z[i];
}

__host__ __device__
dfloat3 cross_product(dfloat3 v1, dfloat3 v2) {
    dfloat3 cross;
    cross.x = v1.y * v2.z - v1.z * v2.y;
    cross.y = v1.z * v2.x - v1.x * v2.z;
    cross.z = v1.x * v2.y - v1.y * v2.x;
    return cross;
}

__host__ __device__
dfloat dot_product(dfloat3 v1, dfloat3 v2) {
    return v1.x * v2.x + v1.y * v2.y + v1.z * v2.z;
}


__host__ __device__
dfloat4 quart_multiplication(dfloat4 q1, dfloat4 q2){
    dfloat4 q;
    
    q.w = q1.w * q2.w - q1.x * q2.x - q1.y * q2.y - q1.z * q2.z;
    q.x = q1.w * q2.x + q1.x * q2.w + q1.y * q2.z - q1.z * q2.y;
    q.y = q1.w * q2.y - q1.x * q2.z + q1.y * q2.w + q1.z * q2.x;
    q.z = q1.w * q2.z + q1.x * q2.y - q1.y * q2.x + q1.z * q2.w;

    return q;
}

__host__ __device__
dfloat3 vector_normalize(dfloat3 v) {
    dfloat inv_length = rsqrtf(v.x * v.x + v.y * v.y + v.z * v.z);
    dfloat3 norm_v;

    if (isnan(inv_length))
        norm_v = dfloat3(0,0,0);
    else
        norm_v = dfloat3(v.x * inv_length, v.y * inv_length, v.z * inv_length);

    return norm_v;
}

__host__ __device__
dfloat4 quart_normalize(dfloat4 q) {
    dfloat norm = sqrt(q.w * q.w + q.x * q.x + q.y * q.y + q.z * q.z);
    q.w /= norm;
    q.x /= norm;
    q.y /= norm;
    q.z /= norm;
    return q;
}

__host__ __device__
dfloat4 quart_addition(dfloat4 q1, dfloat4 q2){
    dfloat4 q;
    q.w = q1.w + q2.w;
    q.x = q1.x + q2.x;
    q.y = q1.y + q2.y;
    q.z = q1.z + q2.z;
    return q;
}

__host__ __device__
dfloat4 vector_to_quart(dfloat3 v) {
    dfloat4 q;
    q.w = 0.0; // The scalar part is zero
    q.x = v.x;
    q.y = v.y;
    q.z = v.z;
    return q;
}

__host__ __device__
void quart_to_rotation_matrix(dfloat4 q, dfloat R[3][3]){
    dfloat qx2 = q.x * q.x;
    dfloat qy2 = q.y * q.y;
    dfloat qz2 = q.z * q.z;
    dfloat qwqx = q.w * q.x;
    dfloat qwqy = q.w * q.y;
    dfloat qwqz = q.w * q.z;
    dfloat qxqy = q.x * q.y;
    dfloat qxqz = q.x * q.z;
    dfloat qyqz = q.y * q.z;

    R[0][0] = 1 - 2 * (qy2 + qz2);
    R[0][1] = 2 * (qxqy - qwqz);
    R[0][2] = 2 * (qxqz + qwqy);

    R[1][0] = 2 * (qxqy + qwqz);
    R[1][1] = 1 - 2 * (qx2 + qz2);
    R[1][2] = 2 * (qyqz - qwqx);

    R[2][0] = 2 * (qxqz - qwqy);
    R[2][1] = 2 * (qyqz + qwqx);
    R[2][2] = 1 - 2 * (qx2 + qy2);
}


// Rotate a vector using a rotation matrix
__host__ __device__
dfloat3 rotate_vector_by_matrix(dfloat3 v, dfloat R[3][3]) {
    dfloat3 v_rot;
    v_rot.x = R[0][0] * v.x + R[0][1] * v.y + R[0][2] * v.z;
    v_rot.y = R[1][0] * v.x + R[1][1] * v.y + R[1][2] * v.z;
    v_rot.z = R[2][0] * v.x + R[2][1] * v.y + R[2][2] * v.z;
    return v_rot;
}

__host__ __device__
dfloat3 rotate_vector_by_quart_R(dfloat3 v, dfloat4 q){
    dfloat R[3][3];
    quart_to_rotation_matrix(q, R);
    return rotate_vector_by_matrix(v, R);
}

// Quaternion conjugate: q_conj = q^*
__host__ __device__
dfloat4 quart_conjugate(dfloat4 q) {
    dfloat4 q_conj;
    q_conj.w = q.w;
    q_conj.x = -q.x;
    q_conj.y = -q.y;
    q_conj.z = -q.z;
    return q_conj;
}

__host__ __device__
dfloat3 rotate_vector_by_quart(dfloat3 v, dfloat4 q) {
    // Convert vector to a quaternion with w = 0
    dfloat4 q_vec =vector_to_quart(v);

    // Calculate rotated quaternion: q_rot = q * q_vec * q^*
    dfloat4 q_conj = quart_conjugate(q);
    dfloat4 q_rot = quart_multiplication(quart_multiplication(q, q_vec), q_conj);

    // The resulting vector is the vector part of q_rot
    dfloat3 v_rot;
    v_rot.x = q_rot.x;
    v_rot.y = q_rot.y;
    v_rot.z = q_rot.z;
    
    return v_rot;
}


// Function to compute the quaternion that rotates vector v1 to vector v2
__host__ __device__
dfloat4 compute_rotation_quart(dfloat3 v1, dfloat3 v2) {
    v1 = vector_normalize(v1);
    v2 = vector_normalize(v2);

    dfloat dot = dot_product(v1, v2);

    // Calculate the angle of rotation
    dfloat angle = acos(dot);

    // Calculate the axis of rotation
    dfloat3 axis = cross_product(v1, v2);
    axis = vector_normalize(axis);

    dfloat4 q;
    q.w = cos(angle / 2.0);
    q.x = axis.x * sin(angle / 2.0);
    q.y = axis.y * sin(angle / 2.0);
    q.z = axis.z * sin(angle / 2.0);

    return q;
}

__host__ __device__
dfloat4 axis_angle_to_quart(dfloat3 axis, dfloat angle) {
    dfloat4 q;
    
    // Normalize the axis of rotation
    axis = vector_normalize(axis);
    
    // Compute the quaternion
    q.w = cos(angle / 2.0);
    q.x = axis.x * sin(angle / 2.0);
    q.y = axis.y * sin(angle / 2.0);
    q.z = axis.z * sin(angle / 2.0);
    
    return q;
}

__host__ __device__
dfloat4 euler_to_quart(dfloat roll, dfloat pitch, dfloat yaw){
    dfloat cr = cos(roll * 0.5);
    dfloat sr = sin(roll * 0.5);
    dfloat cp = cos(pitch * 0.5);
    dfloat sp = sin(pitch * 0.5);
    dfloat cy = cos(yaw * 0.5);
    dfloat sy = sin(yaw * 0.5);

    dfloat4 q;
    q.w = cr * cp * cy + sr * sp * sy;
    q.x = sr * cp * cy - cr * sp * sy;
    q.y = cr * sp * cy + sr * cp * sy;
    q.z = cr * cp * sy - sr * sp * cy;

    return q;
}

__host__ __device__
dfloat3 quart_to_euler(dfloat4 q){
    dfloat3 angles;

    // roll (x-axis rotation)
    double sinr_cosp = 2 * (q.w * q.x + q.y * q.z);
    double cosr_cosp = 1 - 2 * (q.x * q.x + q.y * q.y);
    angles.x = std::atan2(sinr_cosp, cosr_cosp);

    // pitch (y-axis rotation)
    double sinp = std::sqrt(1 + 2 * (q.w * q.y - q.x * q.z));
    double cosp = std::sqrt(1 - 2 * (q.w * q.y - q.x * q.z));
    angles.y = 2 * std::atan2(sinp, cosp) - M_PI / 2;

    // yaw (z-axis rotation)
    double siny_cosp = 2 * (q.w * q.z + q.x * q.y);
    double cosy_cosp = 1 - 2 * (q.y * q.y + q.z * q.z);
    angles.z = std::atan2(siny_cosp, cosy_cosp);

    return angles;
}